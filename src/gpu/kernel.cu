#include "hip/hip_runtime.h"
//The MIT License (MIT)
//Copyright (c) 2016 Massachusetts Institute of Technology

//Authors: Mike Gowanlock
//This software has been created in projects supported by the US National
//Science Foundation and NASA (PI: Pankratius)


//Permission is hereby granted, free of charge, to any person obtaining a copy
//of this software and associated documentation files (the "Software"), to deal
//in the Software without restriction, including without limitation the rights
//to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
//copies of the Software, and to permit persons to whom the Software is
//furnished to do so, subject to the following conditions:
//The above copyright notice and this permission notice shall be included in
//all copies or substantial portions of the Software.
//THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
//IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
//FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
//AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
//LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
//OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN
//THE SOFTWARE.


#include "kernel.h"
#include "structs.h"
#include <math.h>	

/////////////////////////////////////////
//THE RESULTS GET GENERATED AS KEY/VALUE PAIRS IN TWO ARRAYS
//KEY- THE POINT ID BEING SEARCHED
//VALUE- A POINT ID WITHIN EPSILON OF THE KEY POINT THAT WAS SEARCHED
//THE RESULTS ARE SORTED IN SITU ON THE DEVICE BY THRUST AFTER THE KERNEL FINISHES
/////////////////////////////////////////


// __global__ void testkernel(unsigned int * cnt) {
// unsigned int tid=threadIdx.x+ (blockIdx.x*BLOCKSIZE); 
// if (tid>=N2)
// 	return;
	
// unsigned int idx=atomicAdd(cnt,int(1));	
// return;
// }


//this kernel takes as input a direct neighbors table for a greater epsilon value and then calculates the table for a 
//smaller epsilon value 
//N is the total nunber of data points
//lookup is a lookup table that points to indices in the directNeighborArray
//Thats tells where the location of the direct neighbours for each data point are located.
//epsilon is the smaller epsilon value from the lookup table
__global__ void calcNeighborsFromTableKernel(unsigned int *N, struct gpulookuptable * lookup, int * directNeighborArray, unsigned int * cnt, double * epsilon, struct point * database, int * pointIDKey, int * pointInDistVal)
{
unsigned int tid=threadIdx.x+ (blockIdx.x*BLOCKSIZE); 



if (tid>=*N){
	return;
}




// if (tid<*N){
// 	//*cnt=*N;
// 	//*cnt=0;
// 	unsigned int idx=atomicAdd(cnt,int(1));
// }

// return;
//unsigned int idx=atomicAdd(cnt,int(1));



int indexmin=lookup[tid].indexmin;
int indexmax=lookup[tid].indexmax;
double pntX=database[tid].x;
double pntY=database[tid].y;

//uses purely global memory, no optimization here.
for (int i=indexmin; i<=indexmax; i++)
{

	int index=directNeighborArray[i];


	if (sqrt(((pntX-database[index].x)*(pntX-database[index].x))+((pntY-database[index].y)*(pntY-database[index].y)))<=*epsilon)
				{	
					unsigned int idx=atomicAdd(cnt,int(1));
					// results[idx].pointID=tid;
					// results[idx].pointInDist=index;
					pointIDKey[idx]=tid;
					pointInDistVal[idx]=index;


				}		
}





}







//this kernel takes as input a direct neighbors table for a greater epsilon value and then calculates the table for a 
//smaller epsilon value 
//N is the total nunber of data points
//lookup is a lookup table that points to indices in the directNeighborArray
//Thats tells where the location of the direct neighbours for each data point are located.
//epsilon is the smaller epsilon value from the lookup table
__global__ void calcNeighborsFromTableKernelBatches(unsigned int *N, unsigned int *offset, unsigned int *batchNum, struct gpulookuptable * lookup, int * directNeighborArray, unsigned int * cnt, double * epsilon, struct point * database, int * pointIDKey, int * pointInDistVal)
{
unsigned int tid=threadIdx.x+ (blockIdx.x*BLOCKSIZE); 



if (tid>=*N){
	return;
}

unsigned int t_elemID=tid*(*offset)+(*batchNum); //strided across the dataset for more consistent batch return sizes


// if (tid<*N){
// 	//*cnt=*N;
// 	//*cnt=0;
// 	unsigned int idx=atomicAdd(cnt,int(1));
// }

// return;
//unsigned int idx=atomicAdd(cnt,int(1));



int indexmin=lookup[t_elemID].indexmin;
int indexmax=lookup[t_elemID].indexmax;
double pntX=database[t_elemID].x;
double pntY=database[t_elemID].y;

//uses purely global memory, no optimization here.
for (int i=indexmin; i<=indexmax; i++)
{

	int index=directNeighborArray[i];


	if (sqrt(((pntX-database[index].x)*(pntX-database[index].x))+((pntY-database[index].y)*(pntY-database[index].y)))<=*epsilon)
				{	
					unsigned int idx=atomicAdd(cnt,int(1));
					// results[idx].pointID=tid;
					// results[idx].pointInDist=index;
					pointIDKey[idx]=t_elemID;
					pointInDistVal[idx]=index;


				}		
}





}








//kernel with grid to generate the neighbor table for each point in the database
//Each grid cell is assigned to a single block. 
//The threads in each block page the data points to shared memory
//DATA AWARE
//We first see the maximum amount of shared memory required for a single block
//and pass it in to store the overlapping points in adjacent grid cells
__global__ void kernelGridIndexSMBlockDataAware(unsigned int *numThreads, unsigned int *N, unsigned int *debug1, unsigned int *debug2, double *epsilon, struct grid * index, double * gridMin_x, double * gridMin_y, int * gridNumXCells, int * gridNumYCells, int * lookupArr, unsigned int * cnt, struct point * database, const unsigned int * sharedMemElemSize, int * pointIDKey, int * pointInDistVal)
{

unsigned int tid=threadIdx.x+ (blockIdx.x*BLOCKSIZE); 

if (tid>=*numThreads){
	return;
}

//if the cell is empty, then we return, since each cell is processed by a block
if (index[blockIdx.x].indexmin==-1)
{
	return;
}

// *debug1=*sharedMemElemSize;





//stores in shared memory the data points of the originating cells
__shared__ double xOriginCell[BLOCKSIZE];
__shared__ double yOriginCell[BLOCKSIZE];


//testing:
//pool of shared memory for 3 arrays
extern __shared__ double s[];

//using the pool of shared memory defined by s above for the 3 shared memory arrays
//the amount of memory is passed in through the kernel as the 3rd parameter
const int offset1=(*sharedMemElemSize);
const int offset2=2*(*sharedMemElemSize);



double *xAdjacentCell=s;
double *yAdjacentCell=&s[offset1];
int *idPointAdjacentCell=(int *)&s[offset2];




// const unsigned int SMsizeAdjData=*sharedMemElemSize;
// __shared__ double xAdjacentCell[SMsizeAdjData];
// __shared__ double yAdjacentCell[SMsizeAdjData];
// __shared__ int idPointAdjacentCell[SMsizeAdjData];




//the number of adjacent cells with data in them
__shared__ int cellCnt;	
//the maximum number of grid cells around a grid cell is 9 because it's constrained by epsilon
__shared__ int GridCellIDs[9];	



//only one thread calculates the 1D cell IDs of neighboring cells
//this is because each grid cell is assigned to a single block
if (threadIdx.x==0)
{

	//initialize the cell count to 0
	cellCnt=0;

/////////////////////////////
//Calculate the linear ids of the adjacent grid cells of the CELL
//The linearized CELL ID is the block ID
//only those that have points of them are used

	//copy the number of x and y grid cells to registers
	int reg_gridNumXCells=*gridNumXCells;
	int reg_gridNumYCells=*gridNumYCells;	
	

	//int xCellID=(pntX-(*gridMin_x))/(*epsilon);
	//int yCellID=(pntY-(*gridMin_y))/(*epsilon);

	//the block IDx is the 1D linearized grid cell
	int xCellID=blockIdx.x%reg_gridNumXCells;
	int yCellID=blockIdx.x/reg_gridNumXCells;

	

	int minXCellID=0;
	int maxXCellID=0;
	int minYCellID=0;
	int maxYCellID=0;
	

	//calculate the min and max x and y cell ids by adding and subtracting one from each value
	//deal with exception cases below.

	minXCellID=max(0,xCellID-1);
	maxXCellID=min(xCellID+1,reg_gridNumXCells-1);
	minYCellID=max(0,yCellID-1);
	maxYCellID=min(yCellID+1,reg_gridNumYCells-1);

	
	//enumerate the cells in 2D, then convert into 1D
	//only store the cells that have data in them

	
	
	#pragma unroll
	for (int i=minYCellID; i<=maxYCellID; i++){
		#pragma unroll
		for (int j=minXCellID; j<=maxXCellID; j++){
			int linearID=(i*reg_gridNumXCells)+j;			
			
			if(index[linearID].indexmin!=-1) 
			{
				GridCellIDs[cellCnt]=linearID;
				cellCnt++;
			} 
		}
	}


} //end if statement
//End calculate the linear ids of the grid cells
///////////////////////////////


__syncthreads(); //synchronize the threads in the block. Only the first thread in each block has done any work so far



//first, page all of the data elements into shared memory of the adjacent grid cells (and the originating cell itself)
int tmpElemCnt=0;

for (int h=0; h<cellCnt; h++)
{

	int adjCellID=GridCellIDs[h];
	const int numElemInAdjacentCell=index[adjCellID].indexmax-index[adjCellID].indexmin+1;	
	for (int k=0; k<numElemInAdjacentCell; k+=BLOCKSIZE)
	{

		
		if(((k*BLOCKSIZE)+threadIdx.x)<numElemInAdjacentCell)
		{
					int adjDataId=lookupArr[index[adjCellID].indexmin+k+threadIdx.x];	
					int threadOffset=threadIdx.x+k+tmpElemCnt;
					xAdjacentCell[threadOffset]=database[adjDataId].x;
					yAdjacentCell[threadOffset]=database[adjDataId].y;
					idPointAdjacentCell[threadOffset]=adjDataId;
		}

	}

	tmpElemCnt+=numElemInAdjacentCell;
	
}	


__syncthreads(); //sync the threads so that all of the adjacent cell data is in shared memory


//now we compare the adjacent data to the origin cell and perform the distance calculation
const int numElemInOriginCell=index[blockIdx.x].indexmax-index[blockIdx.x].indexmin+1;
	
for (int i=0; i<numElemInOriginCell; i+=BLOCKSIZE)
{
		//manually page data into shared memory		
		//for the origin cell	
		int dataId=lookupArr[index[blockIdx.x].indexmin+i+threadIdx.x];

		
		if(((i*BLOCKSIZE)+threadIdx.x)<numElemInOriginCell)
		{
			xOriginCell[threadIdx.x]=database[dataId].x;
			yOriginCell[threadIdx.x]=database[dataId].y;		
		}

		if(((i*BLOCKSIZE)+threadIdx.x)<numElemInOriginCell)
		{
			//int iterations=min(BLOCKSIZE,(tmpElemCnt-(i*BLOCKSIZE)));
			for (int l=0; l<tmpElemCnt; l++)
			{
				//distance calculation:
				if (sqrt(((xOriginCell[threadIdx.x]-xAdjacentCell[l])*(xOriginCell[threadIdx.x]-xAdjacentCell[l]))+
				((yOriginCell[threadIdx.x]-yAdjacentCell[l])*(yOriginCell[threadIdx.x]-yAdjacentCell[l])))<=(*epsilon))
				{
				unsigned int idx=atomicAdd(cnt,int(1));
				// results[idx].pointID=dataId;
				// results[idx].pointInDist=idPointAdjacentCell[l];
				pointIDKey[idx]=dataId;
				pointInDistVal[idx]=idPointAdjacentCell[l];
				}								
			}
		}

}		




} //end kernel












//kernel with grid to generate the neighbor table for each point in the database
//Each grid cell is assigned to a single block. 
//The threads in each block page the data points to shared memory
//DATA OBLIVIOUS
__global__ void kernelGridIndexSMBlock(unsigned int *numThreads, unsigned int *N, unsigned int *debug1, unsigned int *debug2, double *epsilon, struct grid * index, double * gridMin_x, double * gridMin_y, int * gridNumXCells, int * gridNumYCells, int * lookupArr, unsigned int * cnt, struct point * database, unsigned int * schedule, int * pointIDKey, int * pointInDistVal)
{

unsigned int tid=threadIdx.x+ (blockIdx.x*BLOCKSIZE); 


if (tid>=*numThreads){
	return;
}

//if the cell is empty, then we return, since each cell is processed by a block
// if (index[blockIdx.x].indexmin==-1)
// {
// 	return;
// }


//consult the schedule to find out what grid cell to process:
int cellToProcess=schedule[blockIdx.x];


////////////////////////
//DEBUG
// if (tid==0)
// {
// 	*cnt=*numThreads;
// }

// if (threadIdx.x==0)
// {
// 	atomicAdd(cnt,int(1));
// }
////////////////////////

//the number of adjacent cells with data in them
__shared__ int cellCnt;	
//the maximum number of grid cells around a grid cell is 9 because it's constrained by epsilon
__shared__ int GridCellIDs[9];	

//the number of data items in the originating cell
__shared__ int numElemInOriginCell;


//stores in shared memory the data points of the originating cells
__shared__ double xOriginCell[BLOCKSIZE];
__shared__ double yOriginCell[BLOCKSIZE];

//stores in shared memory the data points of the adjacent cells
__shared__ double xAdjacentCell[BLOCKSIZE];
__shared__ double yAdjacentCell[BLOCKSIZE];
__shared__ int idPointAdjacentCell[BLOCKSIZE];



//only one thread calculates the 1D cell IDs of neighboring cells
//this is because each grid cell is assigned to a single block
if (threadIdx.x==0)
{

	//the number of data items in the originating cell
	numElemInOriginCell=index[cellToProcess].indexmax-index[cellToProcess].indexmin+1;

	//initialize the cell count to 0
	cellCnt=0;

/////////////////////////////
//Calculate the linear ids of the adjacent grid cells of the CELL
//The linearized CELL ID is the block ID
//only those that have points of them are used

	//copy the number of x and y grid cells to registers
	int reg_gridNumXCells=*gridNumXCells;
	int reg_gridNumYCells=*gridNumYCells;	
	

	//int xCellID=(pntX-(*gridMin_x))/(*epsilon);
	//int yCellID=(pntY-(*gridMin_y))/(*epsilon);

	//the block IDx is the 1D linearized grid cell
	int xCellID=cellToProcess%reg_gridNumXCells;
	int yCellID=cellToProcess/reg_gridNumXCells;

	

	int minXCellID=0;
	int maxXCellID=0;
	int minYCellID=0;
	int maxYCellID=0;
	

	//calculate the min and max x and y cell ids by adding and subtracting one from each value
	//deal with exception cases below.

	minXCellID=max(0,xCellID-1);
	maxXCellID=min(xCellID+1,reg_gridNumXCells-1);
	minYCellID=max(0,yCellID-1);
	maxYCellID=min(yCellID+1,reg_gridNumYCells-1);

	
	//enumerate the cells in 2D, then convert into 1D
	//only store the cells that have data in them

	
	
	#pragma unroll
	for (int i=minYCellID; i<=maxYCellID; i++){
		#pragma unroll
		for (int j=minXCellID; j<=maxXCellID; j++){
			int linearID=(i*reg_gridNumXCells)+j;			
			
			if(index[linearID].indexmin!=-1) 
			{
				GridCellIDs[cellCnt]=linearID;
				cellCnt++;
			} 
		}
	}


} //end if statement
//End calculate the linear ids of the grid cells
///////////////////////////////


__syncthreads(); //synchronize the threads in the block. Only the first thread in each block has done any work so far



//loop over each adjacent cell, including the originating cell itself
for (int h=0; h<cellCnt; h++)
{


	 
	//manually page the data of the originating cell into shared memory
	

	for (int i=0; i<numElemInOriginCell; i+=BLOCKSIZE)
	{
			
			
			//manually page data into shared memory		
			//for the origin cell	
			int dataId=lookupArr[index[cellToProcess].indexmin+i+threadIdx.x];

			//if(((i*BLOCKSIZE)+threadIdx.x)<numElemInOriginCell) //CHANGED THIS TO BELOW
			if((i+threadIdx.x)<numElemInOriginCell)
			{
				xOriginCell[threadIdx.x]=database[dataId].x;
				yOriginCell[threadIdx.x]=database[dataId].y;		
			}

			
			//now we page the data of the adjacent cell and perform the distance calculation:

			int adjCellID=GridCellIDs[h];
			const int numElemInAdjacentCell=index[adjCellID].indexmax-index[adjCellID].indexmin+1;


			//do we need a sync threads here? YES
						

			for (int k=0; k<numElemInAdjacentCell; k+=BLOCKSIZE)
			{
				
				

				__syncthreads(); //this one is required


				//Page in data of the adjacent cell:
				//make sure that threads dont page in data that they shouldn't
				//if(((k*BLOCKSIZE)+threadIdx.x)<numElemInAdjacentCell) //CHANGED THIS BELOW
				if((k+threadIdx.x)<numElemInAdjacentCell)
				{
					int adjDataId=lookupArr[index[adjCellID].indexmin+k+threadIdx.x];	
					xAdjacentCell[threadIdx.x]=database[adjDataId].x;
					yAdjacentCell[threadIdx.x]=database[adjDataId].y;
					idPointAdjacentCell[threadIdx.x]=adjDataId;
				}
				

				//do we need a sync threads here? YES
				__syncthreads();

				//the shared memory contains the points of the originating cell and the adjacent cell.
				//each thread processes a single originating cell data point and loops over all of the
				//data points in the adjacent cell

				//make sure that threads dont try to access data out of bounds
				
				//if(((i*BLOCKSIZE)+threadIdx.x)<numElemInOriginCell) //CHANGED THIS TO BELOW
				if((i+threadIdx.x)<numElemInOriginCell) 
				{
					//int iterations=min(BLOCKSIZE,(numElemInAdjacentCell-(k*BLOCKSIZE))); //CHANGED THIS TO BELOW
					int iterations=min(BLOCKSIZE,(numElemInAdjacentCell-k));


					for (int l=0; l<iterations; l++)
					{
						//distance calculation:
						if (sqrt(((xOriginCell[threadIdx.x]-xAdjacentCell[l])*(xOriginCell[threadIdx.x]-xAdjacentCell[l]))+
						((yOriginCell[threadIdx.x]-yAdjacentCell[l])*(yOriginCell[threadIdx.x]-yAdjacentCell[l])))<=(*epsilon))
						{
						unsigned int idx=atomicAdd(cnt,int(1));
						// results[idx].pointID=dataId;
						// results[idx].pointInDist=idPointAdjacentCell[l];
						pointIDKey[idx]=dataId;
						pointInDistVal[idx]=idPointAdjacentCell[l];
						}								
					}
				}
				



			

			} //end adjacent cell paging and distance calculation

			

	}


}//end of outer loop that loops over all of the cells


} //end kernel

















//kernel with grid to generate the neighbor table for each point in the database
//the total number of threads are the number of data points
//each point finds its own surrounding grid cells and compares
//NO SHARED MEMORY

//N-the number of threads
//epislon- the epsilon value to search for between pairs of points
//index- the grid index
//gridMin_x- the minimum x value of the grid
//gridMin_y- the minimum y value of the grid
//gridNumXCells- the number of "x" cells (rows)
//gridNumYCells- the number of "y" cells (columns)
//lookupArr- a look up array that maps the range of ids in the index struct to the data points (database)
//database- the data base of points (their positions)
//offset- an offset into the database of points. Used when batching. Leave it when not batching as well, the offset will be 0.

//produces:
//cnt- global counter of the number of the total number of pairs of points within the the epsilon distance from each other
//results are as key value pairs, where the key is the point being searched, and value is another point within the distance
//pointIDKey- a point within the distance of a point in the pointInDistVal array
//pointInDistVal- a point within the distance of pointIDKey
__global__ void kernelGridIndex(unsigned int *N, unsigned int *offset, unsigned int *batchNum, unsigned int *debug1, unsigned int *debug2, double *epsilon, struct grid * index, double * gridMin_x, double * gridMin_y, int * gridNumXCells, int * gridNumYCells, int * lookupArr, unsigned int * cnt, struct point * database, int * pointIDKey, int * pointInDistVal)
{

unsigned int tid=threadIdx.x+ (blockIdx.x*BLOCKSIZE); 

if (tid>=*N){
	return;
}


//unsigned int t_elemID=tid+(*offset); //original way of batching
unsigned int t_elemID=tid*(*offset)+(*batchNum); //strided across the dataset for more consistent batch return sizes


//copy the x and y positions into registers
	double pntX=database[t_elemID].x;
	double pntY=database[t_elemID].y;

/////////////////////////////
//Calculate the linear ids of the adjacent grid cells of the point
//only those that have points of them are used

	//copy the number of x and y grid cells to registers
	int reg_gridNumXCells=*gridNumXCells;
	int reg_gridNumYCells=*gridNumYCells;	
	

	int xCellID=(pntX-(*gridMin_x))/(*epsilon);
	int yCellID=(pntY-(*gridMin_y))/(*epsilon);

	int GridCellIDs[9];	

	int minXCellID=0;
	int maxXCellID=0;
	int minYCellID=0;
	int maxYCellID=0;
	

	//calculate the min and max x and y cell ids by adding and subtracting one from each value
	//deal with exception cases below.

	minXCellID=max(0,xCellID-1);
	maxXCellID=min(xCellID+1,reg_gridNumXCells-1);
	minYCellID=max(0,yCellID-1);
	maxYCellID=min(yCellID+1,reg_gridNumYCells-1);

	
	//enumerate the cells in 2D, then convert into 1D
	//only store the cells that have data in them
	//the most number of cells that can be found with data is 9 because the grid is constrained by epsilon

	
	int cellCnt=0;	
	#pragma unroll
	for (int i=minYCellID; i<=maxYCellID; i++){
		#pragma unroll
		for (int j=minXCellID; j<=maxXCellID; j++){
			int linearID=(i*reg_gridNumXCells)+j;			
			
			if(index[linearID].indexmin!=-1) 
			{
				GridCellIDs[cellCnt]=linearID;
				cellCnt++;
			} 
		}
	}



//End calculate the linear ids of the grid cells
///////////////////////////////


//iterate over the grid cells with points in them 
for (int j=0; j<cellCnt; j++){
		
		int gridID=GridCellIDs[j];

			
			for (int k=index[gridID].indexmin; k<=index[gridID].indexmax; k++)
			{

				int elemid=lookupArr[k];
				double x2=database[elemid].x;
				double y2=database[elemid].y;	
				if (sqrt(((pntX-x2)*(pntX-x2))+((pntY-y2)*(pntY-y2)))<=(*epsilon))
					{
						unsigned int idx=atomicAdd(cnt,int(1));
						pointIDKey[idx]=t_elemID;
						pointInDistVal[idx]=elemid;
					}


			}

		}	



}



//BATCH ESTIMATOR KERNEL: DOESNT STORE THE RESULTS OR OFFSET
//runs once
//returns cnt
//Samples the total dataset using sampleOffset, so that we don't process the entire database
__global__ void kernelGridIndexBatchEstimator(unsigned int *N, unsigned int *sampleOffset, unsigned int *debug1, unsigned int *debug2, double *epsilon, struct grid * index, double * gridMin_x, double * gridMin_y, int * gridNumXCells, int * gridNumYCells, int * lookupArr, unsigned int * cnt, struct point * database)
{

unsigned int tid=threadIdx.x+ (blockIdx.x*BLOCKSIZE); 

if (tid>=*N){
	return;
}



//copy the x and y positions into registers
	double pntX=database[tid*(*sampleOffset)].x;
	double pntY=database[tid*(*sampleOffset)].y;

/////////////////////////////
//Calculate the linear ids of the adjacent grid cells of the point
//only those that have points of them are used

	//copy the number of x and y grid cells to registers
	int reg_gridNumXCells=*gridNumXCells;
	int reg_gridNumYCells=*gridNumYCells;	
	

	int xCellID=(pntX-(*gridMin_x))/(*epsilon);
	int yCellID=(pntY-(*gridMin_y))/(*epsilon);

	int GridCellIDs[9];	

	int minXCellID=0;
	int maxXCellID=0;
	int minYCellID=0;
	int maxYCellID=0;
	

	//calculate the min and max x and y cell ids by adding and subtracting one from each value
	//deal with exception cases below.

	minXCellID=max(0,xCellID-1);
	maxXCellID=min(xCellID+1,reg_gridNumXCells-1);
	minYCellID=max(0,yCellID-1);
	maxYCellID=min(yCellID+1,reg_gridNumYCells-1);

	
	//enumerate the cells in 2D, then convert into 1D
	//only store the cells that have data in them
	//the most number of cells that can be found with data is 9 because the grid is constrained by epsilon

	
	int cellCnt=0;	
	#pragma unroll
	for (int i=minYCellID; i<=maxYCellID; i++){
		#pragma unroll
		for (int j=minXCellID; j<=maxXCellID; j++){
			int linearID=(i*reg_gridNumXCells)+j;			
			
			if(index[linearID].indexmin!=-1) 
			{
				GridCellIDs[cellCnt]=linearID;
				cellCnt++;
			} 
		}
	}



//End calculate the linear ids of the grid cells
///////////////////////////////


//iterate over the grid cells with points in them 
for (int j=0; j<cellCnt; j++){
		
		int gridID=GridCellIDs[j];

			
			for (int k=index[gridID].indexmin; k<=index[gridID].indexmax; k++)
			{

				int elemid=lookupArr[k];
				double x2=database[elemid].x;
				double y2=database[elemid].y;	
				if (sqrt(((pntX-x2)*(pntX-x2))+((pntY-y2)*(pntY-y2)))<=(*epsilon))
					{
						unsigned int idx=atomicAdd(cnt,int(1));
					}


			}

		}	



}




//kernel with grid to generate the neighbor table for each point in the database
//the total number of threads are the number of data points
//each point finds its own surrounding grid cells and compares
//NO SHARED MEMORY

/////////////////
//FOR TESTING
//////////////////
/*
__global__ void kernelGridIndexKeyVal(unsigned int *N, unsigned int *debug1, unsigned int *debug2, double *epsilon, struct grid * index, double * gridMin_x, double * gridMin_y, int * gridNumXCells, int * gridNumYCells, int * lookupArr, unsigned int * cnt, struct point * database, int * pointIDKey, int * pointInDistVal)
{

unsigned int tid=threadIdx.x+ (blockIdx.x*BLOCKSIZE); 

if (tid>=*N){
	return;
}



//copy the x and y positions into registers
	double pntX=database[tid].x;
	double pntY=database[tid].y;

/////////////////////////////
//Calculate the linear ids of the adjacent grid cells of the point
//only those that have points of them are used

	//copy the number of x and y grid cells to registers
	int reg_gridNumXCells=*gridNumXCells;
	int reg_gridNumYCells=*gridNumYCells;	
	

	int xCellID=(pntX-(*gridMin_x))/(*epsilon);
	int yCellID=(pntY-(*gridMin_y))/(*epsilon);

	int GridCellIDs[9];	

	int minXCellID=0;
	int maxXCellID=0;
	int minYCellID=0;
	int maxYCellID=0;
	

	//calculate the min and max x and y cell ids by adding and subtracting one from each value
	//deal with exception cases below.

	minXCellID=max(0,xCellID-1);
	maxXCellID=min(xCellID+1,reg_gridNumXCells-1);
	minYCellID=max(0,yCellID-1);
	maxYCellID=min(yCellID+1,reg_gridNumYCells-1);

	
	//enumerate the cells in 2D, then convert into 1D
	//only store the cells that have data in them
	//the most number of cells that can be found with data is 9 because the grid is constrained by epsilon

	
	int cellCnt=0;	
	#pragma unroll
	for (int i=minYCellID; i<=maxYCellID; i++){
		#pragma unroll
		for (int j=minXCellID; j<=maxXCellID; j++){
			int linearID=(i*reg_gridNumXCells)+j;			
			
			if(index[linearID].indexmin!=-1) 
			{
				GridCellIDs[cellCnt]=linearID;
				cellCnt++;
			} 
		}
	}




//End calculate the linear ids of the grid cells
///////////////////////////////


//iterate over the grid cells with points in them 
for (int j=0; j<cellCnt; j++){
		
		int gridID=GridCellIDs[j];

			
			for (int k=index[gridID].indexmin; k<=index[gridID].indexmax; k++)
			{

				int elemid=lookupArr[k];
				double x2=database[elemid].x;
				double y2=database[elemid].y;	
				if (sqrt(((pntX-x2)*(pntX-x2))+((pntY-y2)*(pntY-y2)))<=(*epsilon))
					{
						unsigned int idx=atomicAdd(cnt,int(1));
						// results[idx].pointID=tid;
						// results[idx].pointInDist=elemid;
						
						pointIDKey[idx]=tid;
						pointInDistVal[idx]=elemid;
					}


			}

		}	



}


*/




//Kernel brute forces to generate the neighbor table for each point in the database
__global__ void kernelBruteForce(unsigned int *N, unsigned int *debug1, unsigned int *debug2, double *epsilon, unsigned int * cnt, struct point * database, int * pointIDKey, int * pointInDistVal) {

unsigned int tid=threadIdx.x+ (blockIdx.x*BLOCKSIZE); 

if (tid>=*N){
	return;
}


// if (tid==0)
// {
// 	*debug1=555;
// }

double pntX=database[tid].x;
double pntY=database[tid].y;

//original, only use global memory
// for (int i=0; i<N; i++)
// {
// 	if (sqrt(((pntX-a[i])*(pntX-a[i]))+((pntY-b[i])*(pntY-b[i])))<=*epsilon)
// 	{
		
// 		atomicAdd(cnt,int(1));
// 	}	
// }


__shared__ double x[BLOCKSIZE];
__shared__ double y[BLOCKSIZE];





//three seperate loops for the "LEFTOVERS" to avoid divergent branching
//The first two relate to those threads that make up full blocks that compare against each other
//and partial blocks, and those threads that make up a partial block that need to compare against
//the rest of the data
int iterations1=(*N/BLOCKSIZE)*BLOCKSIZE;
int iterations2=*N%BLOCKSIZE;

// if (tid==0)
// {
// *debug1=iterations1;
// *debug2=iterations2;
	
// }

__syncthreads();



//for those threads that are in full blocks
if (tid<iterations1)
	{

	//for (int i=0; i<1; i++)
	for (int i=0; i<iterations1; i+=BLOCKSIZE)
	{
		
		
		
		//manually page data into shared memory with coalescing
		int elemid=i+threadIdx.x;
		// x[threadIdx.x]=a[elemid];
		// y[threadIdx.x]=b[elemid];
		x[threadIdx.x]=database[elemid].x;
		y[threadIdx.x]=database[elemid].y;
		__syncthreads();
		
		//#pragma unroll
		for (int j=0; j<BLOCKSIZE; j++)
		{
				unsigned int idxx=atomicAdd(debug1,int(1));
				//global mem
				//if (sqrt(((pntX-a[i+j])*(pntX-a[i+j]))+((pntY-b[i+j])*(pntY-b[i+j])))<=*epsilon)
				//shared mem
				if (sqrt(((pntX-x[j])*(pntX-x[j]))+((pntY-y[j])*(pntY-y[j])))<=*epsilon)
				{	
					unsigned int idx=atomicAdd(cnt,int(1));
					//resultset[idx]=i+j; //
					//resultsetElem[idx]=tid; //
					// results[idx].pointID=tid;
					// results[idx].pointInDist=i+j;

					pointIDKey[idx]=tid;
					pointInDistVal[idx]=i+j;

				}		
			
		}
		__syncthreads();
	}


	//START LEFTOVERS (from the nice tiled calculations)

	//manually page data into shared memory with coalescing
	if (threadIdx.x<iterations2)
	{
	int elemid=iterations1+threadIdx.x;	
	// x[threadIdx.x]=a[elemid];
	// y[threadIdx.x]=b[elemid];
	x[threadIdx.x]=database[elemid].x;
	y[threadIdx.x]=database[elemid].y;
	}
	__syncthreads();

	//for (int j=0; j<iterations2; j++)
	//for (int j=iterations1; j<iterations1+iterations2; j++) //for global memory
	for (int j=0; j<iterations2; j++)
	{
			unsigned int idxx=atomicAdd(debug1,int(1));
			//access directly in global memory
			//if (sqrt(((pntX-a[j])*(pntX-a[j]))+((pntY-b[j])*(pntY-b[j])))<=*epsilon)
			
			if (sqrt(((pntX-x[j])*(pntX-x[j]))+((pntY-y[j])*(pntY-y[j])))<=*epsilon)
			{	
				unsigned int idx=atomicAdd(cnt,int(1));
				// results[idx].pointID=tid;
				// results[idx].pointInDist=iterations1+j;

				pointIDKey[idx]=tid;
				pointInDistVal[idx]=iterations1+j;
			}		

	}

	//END LEFTOVERS (for nice tiled calculations)

} //end of nice tiled calculations


//start of the "leftover" threads that don't make up a full block
// if (tid>=iterations1)
// {
// 	for (int j=0; j<N; j++)
// 	{
// 		if (sqrt(((pntX-a[j])*(pntX-a[j]))+((pntY-b[j])*(pntY-b[j])))<=*epsilon)
// 		{
// 			atomicAdd(cnt,int(1));
// 		}
// 	}
// }

//start of the "leftover" threads that don't make up a full block
//they have to compare themselves to the rest of the data

if (tid>=iterations1)
{
	
	for (int i=0; i<*N; i+=iterations2)
	{

	
		if (threadIdx.x<iterations2)
		{
		int elemid=i+threadIdx.x;	
		// x[threadIdx.x]=a[elemid];
		// y[threadIdx.x]=b[elemid];
		x[threadIdx.x]=database[elemid].x;
		y[threadIdx.x]=database[elemid].y;
		}
		__syncthreads();

		int nextIterations=min(iterations2,*N-i);

		for (int j=0; j<nextIterations; j++)
		{
			unsigned int idxx=atomicAdd(debug1,int(1));
			//if (sqrt(((pntX-a[j])*(pntX-a[j]))+((pntY-b[j])*(pntY-b[j])))<=*epsilon)
			if (sqrt(((pntX-x[j])*(pntX-x[j]))+((pntY-y[j])*(pntY-y[j])))<=*epsilon)
			{
				unsigned int idx=atomicAdd(cnt,int(1));
				// results[idx].pointID=tid;
				// results[idx].pointInDist=i+j;

				pointIDKey[idx]=tid;
				pointInDistVal[idx]=i+j;


			}
		}

		__syncthreads();

	} //end loop	

} //end of if


return;
}