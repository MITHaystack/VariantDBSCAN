#include "hip/hip_runtime.h"
//The MIT License (MIT)
//Copyright (c) 2016 Massachusetts Institute of Technology

//Authors: Mike Gowanlock
//This software has been created in projects supported by the US National
//Science Foundation and NASA (PI: Pankratius)


//Permission is hereby granted, free of charge, to any person obtaining a copy
//of this software and associated documentation files (the "Software"), to deal
//in the Software without restriction, including without limitation the rights
//to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
//copies of the Software, and to permit persons to whom the Software is
//furnished to do so, subject to the following conditions:
//The above copyright notice and this permission notice shall be included in
//all copies or substantial portions of the Software.
//THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
//IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
//FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
//AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
//LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
//OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN
//THE SOFTWARE.

#include <cstdlib>
#include <stdio.h>
#include <random>
#include "prototypes.h"
#include "globals.h"
#include "omp.h"
#include "DBScan.h"
#include "schedule.h"
#include <algorithm> 
#include <string.h>
#include <fstream>
#include <iostream>
#include <string>
#include "GPU.h"
#include "kernel.h"
#include "cluster.h"
#include <math.h>
#include <queue>
#include <iomanip>

//#include "structs.h"
using namespace std;





void generateGridDimensions(std::vector<struct dataElem>* dataPoints, double epsilon, double * gridMin_x, double * gridMin_y, int * gridNumXCells, int * gridNumYCells);
void populateGridIndexAndLookupArray(std::vector<struct dataElem>* dataPoints, double epsilon, struct grid * index, unsigned int * lookupArr, double gridMin_x, double gridMin_y, int gridNumXCells, int gridNumYCells, int * numFullCells);
void calcLinearID(struct dataElem * point, struct grid * index, double epsilon, double gridMin_x, double gridMin_y, int gridNumXCells, int gridNumYCells, int * outGridCellIDs, int * outNumGridCells);

//calculates the maximum number of points that will be required by any originating grid cell
unsigned int calcMaxSharedMemDataAware(struct grid * index, int gridNumXCells, int gridNumYCells);


//NOTE:
//IN ALL IMPLEMENTATIONS, WE ONLY INDEX DATA BASED ON 2 DIMENSIONS, LAT AND LONG. HOWEVER, THE TIME, AND TEC VALUES ARE STILL IMPORTED AS DATA ELEMENTS FOR POST-PROCESSING

int main(int argc, char *argv[])
{
	


	/////////////////////////
	// Get information from command line
	//1) the dataset, 2) list of parameter values for experiments.
	//The dataset name is stored in: inputFname, and the experiment instances are stored in a struct containing epsilon and minpts (experimentList)
	/////////////////////////

	//Read in parameters from file:
	//dataset filename and cluster instance file
	if (argc!=3)
	{
	cout <<"\n\nIncorrect number of input parameters.  \nShould be dataset file, and DBScan instance file (outlines the experiments)\n";
	return 0;
	}
	
	//copy parameters from commandline:
	//char inputFname[]="data/test_data_removed_nan.txt";	
	char inputFname[500];
	char inputInstanceName[500];

	strcpy(inputFname,argv[1]);
	strcpy(inputInstanceName,argv[2]);

	printf("\nDataset file: %s",inputFname);
	printf("\nExperiment file: %s",inputInstanceName);

	

	//make a vector that stores the experiments
	std::vector<struct experiment> experimentList;
	importDBScanInstances(&experimentList, inputInstanceName);








	//////////////////////////////
	//import the dataset:
	/////////////////////////////
	std::vector<struct dataElem> dataPoints;
	

	//char inputFname[]="data/test_data_removed_nan.txt";
	importDataset(&dataPoints, inputFname);
	
	


	//sort the data in the following order: x,y,TEC value, time
	//qsort((void *) &dataPoints,dataPoints.size(),sizeof(struct dataElem),(compfn)compareDataElemStructFunc); 
	std::sort(dataPoints.begin(),dataPoints.end(),compareDataElemStructFunc);

	

	
	//experiment parameters (temporary)
	// double * epsilon;
	// epsilon=(double*)malloc(sizeof(double));
	// *epsilon=experimentList[0].epsilon;

	// int * minpts;
	// minpts=(int*)malloc(sizeof(int));
	// *minpts=experimentList[0].minpts;

	



	//////////////////////////////////////
	//GENERATE THE GRID INDEX FOR THE GPU FOR EACH VALUE OF EPSILON
	//////////////////////////////////////

	


	printf("\n\nCOMMENTED ORIGINAL GRID INDEX!!!\n\n");

	/*
	//double GPUGridTstart=omp_get_wtime();
	double gridMin_x=0;
	double gridMin_y=0; 
	int gridNumXCells=0; 
	int gridNumYCells=0;

	//generate grid dimensions:
	generateGridDimensions(&dataPoints, *epsilon, &gridMin_x, &gridMin_y, &gridNumXCells, &gridNumYCells);
	// printf("\n In main: Min x: %f, Min y: %f",gridMin_x,gridMin_y);
	printf("\n In main: Min x: %f, Min y: %f, Num X cells: %d, Num Y cells: %d",gridMin_x,gridMin_y,gridNumXCells,gridNumYCells);
	

	//Allocate grid index and lookup array
	struct grid * index;
	index=new grid[gridNumXCells*gridNumYCells];

	unsigned int * lookupArr;
	lookupArr=new unsigned int[dataPoints.size()];

	
	//number of full cells:
	int * numFullCells;
	numFullCells=(int*)malloc(sizeof(int));
	*numFullCells=0;


	//populate the index
	populateGridIndexAndLookupArray(&dataPoints, *epsilon, index, lookupArr, gridMin_x, gridMin_y, gridNumXCells, gridNumYCells, numFullCells);
	double GPUGridTend=omp_get_wtime();
	double totalGPUGridTtotal=GPUGridTend - GPUGridTstart;



	printf("\ntime to populate the grid and lookup array: %f",totalGPUGridTtotal);
	printf("\nIn main number of full cells: %d", *numFullCells);
	*/



	///////////////
	//BRUTE FORCE GPU
	//NO BATCHING
	///////////////
	#if SEARCHMODE==0


	double tstart_bruteforcegpu=omp_get_wtime();

	//neighbor table:
	table * neighborTable;
	neighborTable=new table[dataPoints.size()];
	int * totalNeighbors;
	totalNeighbors=(int*)malloc(sizeof(int));
	*totalNeighbors=0;

	printf("\nBrute force GPU (NO BATCHING):");
	

	double tstart=omp_get_wtime();
	makeDistanceTableGPUBruteForce(&dataPoints,epsilon, neighborTable, totalNeighbors);
	double tend=omp_get_wtime();
	printf("\nBRUTE FORCE Time on GPU: %f",tend-tstart);cout.flush();
	printf("\nTotal neighbours in table: %d", *totalNeighbors);

	double dbscantstartgpuindex=omp_get_wtime();
	dbscan(neighborTable, dataPoints.size(), *minpts); //5 is minpts
	double dbscantendgpuindex=omp_get_wtime();
	printf("\ntime to dbscan: %f",dbscantendgpuindex-dbscantstartgpuindex);

	double tend_bruteforcegpu=omp_get_wtime();
	printf("\nTotal time GPU brute force: %f", tend_bruteforcegpu - tstart_bruteforcegpu);

	printf("\n*********************************");
	
	#endif

	//////////////
	//END GPU BRUTE FORCE
	//NO BATCHING
	//////////////



	///////////////
	//GLOBAL MEMORY GRID KERNEL
	//NO BATCHING
	///////////////

	#if SEARCHMODE==1

	printf("\nSINGLE EXECUTION OF THE KERNEL (GLOBAL MEMORY) AND DBSCAN.  ONLY USED TO EVALUATE KERNEL PERFORMANCE USING THE PROFILER(no batching).");

	//SET UP THE INDEX:

	double gridMin_x=0;
	double gridMin_y=0; 
	int gridNumXCells=0; 
	int gridNumYCells=0;

	//generate grid dimensions:
	generateGridDimensions(&dataPoints, experimentList[0].epsilon, &gridMin_x, &gridMin_y, &gridNumXCells, &gridNumYCells);
	// printf("\n In main: Min x: %f, Min y: %f",gridMin_x,gridMin_y);
	printf("\n In main: Min x: %f, Min y: %f, Num X cells: %d, Num Y cells: %d",gridMin_x,gridMin_y,gridNumXCells,gridNumYCells);
	

	//Allocate grid index and lookup array
	struct grid * index;
	index=new grid[gridNumXCells*gridNumYCells];

	unsigned int * lookupArr;
	lookupArr=new unsigned int[dataPoints.size()];

	
	//number of full cells:
	int * numFullCells;
	numFullCells=(int*)malloc(sizeof(int));
	*numFullCells=0;


	//populate the index
	double GPUGridTstart=omp_get_wtime();
	populateGridIndexAndLookupArray(&dataPoints, experimentList[0].epsilon, index, lookupArr, gridMin_x, gridMin_y, gridNumXCells, gridNumYCells, numFullCells);
	double GPUGridTend=omp_get_wtime();
	double totalGPUGridTtotal=GPUGridTend - GPUGridTstart;



	printf("\ntime to populate the grid and lookup array: %f",totalGPUGridTtotal);
	printf("\nIn main number of full cells: %d", *numFullCells);


	//END SET UP THE INDEX


	//trials:

	for(int h=0; h<NUM_TRIALS; h++)
	{




		printf("\n\nTRIAL NUM: %d", h);


		double totalGPUTstart=omp_get_wtime();

		int * totalNeighborsGPU;
		totalNeighborsGPU=(int*)malloc(sizeof(int));
		*totalNeighborsGPU=0;		

		table * neighborTable;
		neighborTable=new table[dataPoints.size()];


		printf("\nCalling GPU grid kernel\n WITH NOSHARED MEMORY KERNEL\n Single batch\n");
		
		
		double gpuGridTstart=omp_get_wtime();
		makeDistanceTableGPUGridIndex(&dataPoints, &experimentList[0].epsilon, index, &gridMin_x, &gridMin_y, &gridNumXCells, &gridNumYCells, lookupArr, neighborTable, totalNeighborsGPU);	
		double gpuGridTend=omp_get_wtime();
		
		printf("\nTime to run the GPU Grid table implementation: %f", gpuGridTend - gpuGridTstart);
		
		printf("\nEnd calling GPU grid kernel");

		double dbscantstartgpuindex=omp_get_wtime();
		dbscan(neighborTable, dataPoints.size(), experimentList[0].minpts); //5 is minpts
		double dbscantendgpuindex=omp_get_wtime();
		printf("\ntime to dbscan: %f",dbscantendgpuindex-dbscantstartgpuindex);

		double totalGPUTend=omp_get_wtime();

		printf("\nTime excluding the grid construction and lookup array: %f", totalGPUTend-totalGPUTstart);
		printf("\nGrand total time gpu dbscan with indexing: %f", (totalGPUTend - totalGPUTstart) + totalGPUGridTtotal);

	}


	#endif

	///////////////
	//END GLOBAL MEMORY GRID KERNEL
	//NO BATCHING
	///////////////




	///////////////
	//SHARED MEMORY GRID KERNEL
	//DATA OBLIVIOUS- TILE THE COMPUTATION
	//NO BATCHING
	///////////////
	#if SEARCHMODE==2

	printf("\nSINGLE EXECUTION OF THE KERNEL (SHARED MEMORY) AND DBSCAN.  ONLY USED TO EVALUATE KERNEL PERFORMANCE USING THE PROFILER(no batching).");


	double GPUGridTstart=omp_get_wtime();
	double gridMin_x=0;
	double gridMin_y=0; 
	int gridNumXCells=0; 
	int gridNumYCells=0;

	//generate grid dimensions:
	generateGridDimensions(&dataPoints, experimentList[0].epsilon, &gridMin_x, &gridMin_y, &gridNumXCells, &gridNumYCells);
	// printf("\n In main: Min x: %f, Min y: %f",gridMin_x,gridMin_y);
	printf("\n In main: Min x: %f, Min y: %f, Num X cells: %d, Num Y cells: %d",gridMin_x,gridMin_y,gridNumXCells,gridNumYCells);
	

	//Allocate grid index and lookup array
	struct grid * index;
	index=new grid[gridNumXCells*gridNumYCells];

	unsigned int * lookupArr;
	lookupArr=new unsigned int[dataPoints.size()];

	
	//number of full cells:
	int * numFullCells;
	numFullCells=(int*)malloc(sizeof(int));
	*numFullCells=0;


	//populate the index
	populateGridIndexAndLookupArray(&dataPoints, experimentList[0].epsilon, index, lookupArr, gridMin_x, gridMin_y, gridNumXCells, gridNumYCells, numFullCells);
	double GPUGridTend=omp_get_wtime();
	double totalGPUGridTtotal=GPUGridTend - GPUGridTstart;



	printf("\ntime to populate the grid and lookup array: %f",totalGPUGridTtotal);
	printf("\nIn main number of full cells: %d", *numFullCells);




	

	//trials:

	for(int h=0; h<NUM_TRIALS; h++)
	{




		printf("\n\nTRIAL NUM: %d", h);


		double totalGPUTstart=omp_get_wtime();


		int * totalNeighborsGPU;
		totalNeighborsGPU=(int*)malloc(sizeof(int));
		*totalNeighborsGPU=0;

		table * neighborTable;
		neighborTable=new table[dataPoints.size()];


		printf("\nCalling GPU grid kernel With Shared Memory\n DATA OBLIVIOUS\nEach non-empty grid cell processed by a block!\nSingle Batch\n");
		double gpuGridTstart=omp_get_wtime();
		makeDistanceTableGPUGridIndexWithSMBlockDataOblivious(&dataPoints, &experimentList[0].epsilon, index, numFullCells, &gridMin_x, &gridMin_y, &gridNumXCells, &gridNumYCells, lookupArr, neighborTable, totalNeighborsGPU);
		double gpuGridTend=omp_get_wtime();
		


		printf("\nTime to run the GPU Grid table implementation: %f", gpuGridTend - gpuGridTstart);
		
		printf("\nEnd calling GPU grid kernel");




	double dbscantstartgpuindex=omp_get_wtime();
	dbscan(neighborTable, dataPoints.size(), experimentList[0].minpts); 
	double dbscantendgpuindex=omp_get_wtime();
	printf("\ntime to dbscan: %f",dbscantendgpuindex-dbscantstartgpuindex);


	double totalGPUTend=omp_get_wtime();

	printf("\nTime excluding the grid construction and lookup array: %f", totalGPUTend-totalGPUTstart);
	printf("\nGrand total Time gpu dbscan with indexing: %f", (totalGPUTend - totalGPUTstart) + totalGPUGridTtotal);
	
	}

	

	#endif

	///////////////
	//END SHARED MEMORY GRID KERNEL
	//DATA OBLIVIOUS- TILE THE COMPUTATION
	//NO BATCHING
	///////////////



	///////////////
	//GRID KERNEL GLOBAL MEMORY
	//BATCHING
	///////////////
	#if SEARCHMODE==3

	printf("\nCalling GPU grid kernel\n WITH NOSHARED MEMORY KERNEL\n");
	double totalGPUTstart=omp_get_wtime();

	int * totalNeighborsGPU;
	totalNeighborsGPU=(int*)malloc(sizeof(int));
	*totalNeighborsGPU=0;		

	table * neighborTable;
	neighborTable=new table[dataPoints.size()];

	
	double gpuGridTstart=omp_get_wtime();
	makeDistanceTableGPUGridIndexBatches(&dataPoints,epsilon, index, &gridMin_x, &gridMin_y, &gridNumXCells, &gridNumYCells, lookupArr, neighborTable, totalNeighborsGPU);
	double gpuGridTend=omp_get_wtime();
	
	printf("\nTime to run the GPU Grid table implementation: %f", gpuGridTend - gpuGridTstart);
	
	printf("\nEnd calling GPU grid kernel");

	// printf("\nEXITING early...");
	// return 0;

	double dbscantstartgpuindex=omp_get_wtime();
	dbscan(neighborTable, dataPoints.size(), *minpts); //5 is minpts
	double dbscantendgpuindex=omp_get_wtime();
	printf("\nTime to dbscan: %f",dbscantendgpuindex-dbscantstartgpuindex);

	double totalGPUTend=omp_get_wtime();

	printf("\nTime excluding the grid construction and lookup array: %f", totalGPUTend-totalGPUTstart);
	printf("\nGrand total Time gpu dbscan with indexing: %f", (totalGPUTend - totalGPUTstart) + totalGPUGridTtotal);


	#endif




	///////////////////////////////////////////
	//GENERATING ONE TABLE FROM ANOTHER ON THE GPU
	///////////////////////////////////////////
	#if SEARCHMODE==4


	printf("\nGENERATING ONE TABLE FROM ANOTHER ONE THE GPU\n");
	printf("\nThis is an example using the Single Batch Implementation");

	
	
	//a lookup table that points to the array of neighbors above
	//because we cant use vectors on the GPU
	struct gpulookuptable * gpuLookupArray;
	gpuLookupArray= new gpulookuptable[dataPoints.size()];
	


	//first neighbor table:
	//generate the second table from the first table
	//neighbor table:
	table * neighborTable;
	neighborTable=new table[dataPoints.size()];
	

	int * totalNeighborsGPU;
	totalNeighborsGPU=(int*)malloc(sizeof(int));
	*totalNeighborsGPU=0;		




	
	
	double gpuGridTstart=omp_get_wtime();
	makeDistanceTableGPUGridIndex(&dataPoints,epsilon, index, &gridMin_x, &gridMin_y, &gridNumXCells, &gridNumYCells, lookupArr, neighborTable, totalNeighborsGPU);	
	double gpuGridTend=omp_get_wtime();
	
	printf("\nTime to run the GPU Grid table implementation: %f", gpuGridTend - gpuGridTstart);
	

	double dbscantstartgpuindex=omp_get_wtime();
	dbscan(neighborTable, dataPoints.size(), *minpts); //5 is minpts
	double dbscantendgpuindex=omp_get_wtime();
	printf("\ntime to dbscan: %f",dbscantendgpuindex-dbscantstartgpuindex);

	double totalGPUTend=omp_get_wtime();

	

	


	//second neighbor table:
	//generate the second table from the first table
	//neighbor table:

	printf("\n*****************\nGenerating subsequent table:");
	int * directNeighborArray;
	directNeighborArray=new int[*totalNeighborsGPU];

	generateNeighborArrayForGPU(dataPoints.size(),neighborTable, directNeighborArray, gpuLookupArray);	

	table * neighborTable2;
	neighborTable2=new table[dataPoints.size()];


	double * epsilon2;
	epsilon2=(double*)malloc(sizeof(double));
	*epsilon2=*epsilon/2.0;

	printf("\nHalf of epsilon: %f",*epsilon2);

	double tstart2=omp_get_wtime();
	generateDistanceTableFromPreviousTable(&dataPoints, gpuLookupArray, directNeighborArray, totalNeighborsGPU, epsilon2, neighborTable2);
	double tend2=omp_get_wtime();
	printf("\nTime to create subsequent table for second epsilon value %f: %f",*epsilon2,tend2-tstart2);

	//////////////////////////////////



	//call DBSCAN (from first table- neighborTable):
	double dbscantstart=omp_get_wtime();
	dbscan(neighborTable2, dataPoints.size(), *minpts); //5 is minpts
	double dbscantend=omp_get_wtime();
	printf("\ntime to dbscan: %f",dbscantend-dbscantstart);

	
	



	#endif





	///////////////
	//GRID KERNEL GLOBAL MEMORY
	//BATCHING-- DBSCAN WITH A PREVIOUSLY GENERATED NEIGHBOR TABLE
	//ONE ONE INITIAL TABLE FOR THE LARGEST EPSILON
	//THE REST OF THE VARIANTS CLUSTER FROM THE FIRST ONE BY FILTERING THE CANDIDATES
	///////////////
	#if SEARCHMODE==5

	printf("\nCalling GPU grid kernel\n WITH NOSHARED MEMORY KERNEL\n");
	printf("\n***DBSCAN THAT FILTERS FROM A PREVIOUS NEIGHBORTABLE FOR SMALLER EPSILON***\n");
	double totalGPUTstart=omp_get_wtime();

	int * totalNeighborsGPU;
	totalNeighborsGPU=(int*)malloc(sizeof(int));
	*totalNeighborsGPU=0;		

	table * neighborTable;
	neighborTable=new table[dataPoints.size()];

	

	printf("\nGenerating the NeighborTable for the largest epsilon value: %f", experimentList[0].epsilon);	
	double gpuGridTstart=omp_get_wtime();
	makeDistanceTableGPUGridIndexBatches(&dataPoints, &experimentList[0].epsilon, index, &gridMin_x, &gridMin_y, &gridNumXCells, &gridNumYCells, lookupArr, neighborTable, totalNeighborsGPU);
	double gpuGridTend=omp_get_wtime();
	
	printf("\nTime to run the GPU Grid table implementation: %f", gpuGridTend - gpuGridTstart);
	
	printf("\nEnd calling GPU grid kernel");


	//dbscan all of the instances using the neighbortable
	//the first instance dbscans with the dbscan method because all of the neighbors are direct neighbors and don't
	//need to be filtered
	//The rest need to call DBScan with a filter for the additional points
	int nthreads=std::min((int)experimentList.size(),16);
	omp_set_num_threads(nthreads);
	#pragma omp parallel for 
	for (int i=0; i<experimentList.size(); i++)
	{
		double dbscantstartgpuindex=omp_get_wtime();	
		if (i==0){
			dbscan(neighborTable, dataPoints.size(), experimentList[i].minpts); 
		}
		else
		{
			dbscanWithFilter(&dataPoints, neighborTable, experimentList[i].epsilon, experimentList[i].minpts);		
		}

		double dbscantendgpuindex=omp_get_wtime();	
		printf("\nTime to dbscan: %f with Epsilon: %f, minpts: %d",dbscantendgpuindex-dbscantstartgpuindex, experimentList[i].epsilon, experimentList[i].minpts);
	}
	

	// double dbscantstartgpuindex=omp_get_wtime();
	// dbscan(neighborTable, dataPoints.size(), *minpts); //5 is minpts
	// double dbscantendgpuindex=omp_get_wtime();
	// printf("\nTime to dbscan: %f",dbscantendgpuindex-dbscantstartgpuindex);


	// double * epsilon2;
	// epsilon2=(double*)malloc(sizeof(double));
	// *epsilon2=*epsilon/2.0;

	// printf("\nNow running DBSCAN (with filtering) for half of the epsilon value: %f", *epsilon2);
	// double dbscantstartgpuindex2=omp_get_wtime();
	// dbscanWithFilter(dataPoints, neighborTable, dataPoints.size(), *epsilon2, *minpts);
	// double dbscantendgpuindex2=omp_get_wtime();
	// printf("\nTime to dbscan second: %f",dbscantendgpuindex2-dbscantstartgpuindex2);
	

	double totalGPUTend=omp_get_wtime();

	printf("\nTime excluding the grid construction and lookup array: %f", totalGPUTend-totalGPUTstart);
	printf("\nGrand total Time gpu dbscan with indexing: %f", (totalGPUTend - totalGPUTstart) + totalGPUGridTtotal);


	#endif


	///////////////
	//GRID KERNEL GLOBAL MEMORY
	//BATCHING-- DBSCAN WITH A PREVIOUSLY GENERATED NEIGHBOR TABLE
	//ONE TABLE PER DBSCAN VARIANT
	//EITHER GENERATE THE NEIGHBORTABLE USING THE INDEX, OR FROM A PREVIOUS TABLE
	//DEPRECATED, USING THE NEW NEIGHBORTABLES
	///////////////
	#if SEARCHMODE==6

	printf("\nCalling GPU grid kernel\n WITH NOSHARED MEMORY KERNEL\n");
	printf("\n***DBSCAN THAT GENERATES A NEIGHBORTABLE FOR EACH VARIANT INSTANCE***\n");

	printf("\n*******\nCreating Indexes for each experiment, although we may not use all of them.\n\n");

	double pipeline_GPUGridTstart=omp_get_wtime();

	double * pipeline_gridMin_x;
	pipeline_gridMin_x=new double[experimentList.size()];

	double * pipeline_gridMin_y;
	pipeline_gridMin_y=new double[experimentList.size()]; 

	int * pipeline_gridNumXCells;
	pipeline_gridNumXCells= new int [experimentList.size()]; 
	
	int * pipeline_gridNumYCells;
	pipeline_gridNumYCells=new int [experimentList.size()];

	

	//pointers to the grid indexes for each experiment
	//the memory is allocated in the loop below once then umber of cells have been calculated for each experiment
	struct grid ** pipeline_index=new grid*[experimentList.size()];
	
	
	
	//Allocate lookup array
	unsigned int ** pipeline_lookupArr=new unsigned int*[experimentList.size()];
	for (int i=0; i<experimentList.size(); i++)
	{
	pipeline_lookupArr[i]=new unsigned int[dataPoints.size()];
	}
	
	//number of full cells:
	int * pipeline_numFullCells;
	pipeline_numFullCells=new int[experimentList.size()];

	//initialize:
	for (int i=0; i<experimentList.size(); i++)
	{
		pipeline_numFullCells[i]=0;
	}	

	
	

	for (int i=0; i<experimentList.size(); i++)
	{
	printf("\n**************\nCreating index for the experiment epsilon: %f, minpts: %d ",experimentList[i].epsilon,experimentList[i].minpts);	
	//generate grid dimensions:
	generateGridDimensions(&dataPoints, experimentList[i].epsilon, &pipeline_gridMin_x[i], &pipeline_gridMin_y[i], &pipeline_gridNumXCells[i], &pipeline_gridNumYCells[i]);
	
	//allocate memory for the index now that the number of cells have been calculated.
	unsigned int numcells=pipeline_gridNumXCells[i]*pipeline_gridNumYCells[i];
	pipeline_index[i]=new grid[numcells];

	// printf("\n In main: Min x: %f, Min y: %f",gridMin_x,gridMin_y);
	printf("\n Populating indexes for the pipeline: Min x: %f, Min y: %f, Num X cells: %d, Num Y cells: %d",pipeline_gridMin_x[i],pipeline_gridMin_y[i],pipeline_gridNumXCells[i],pipeline_gridNumYCells[i]);
	//populate the index
	populateGridIndexAndLookupArray(&dataPoints, experimentList[i].epsilon, pipeline_index[i], pipeline_lookupArr[i], pipeline_gridMin_x[i], pipeline_gridMin_y[i], pipeline_gridNumXCells[i], pipeline_gridNumYCells[i], &pipeline_numFullCells[i]);
	printf("\nIn main number of full cells: %d", pipeline_numFullCells[i]);cout.flush();
	}

	double pipeline_GPUGridTend=omp_get_wtime();
	

	double pipeline_totalGPUGridTtotal=pipeline_GPUGridTend - pipeline_GPUGridTstart;

	printf("\ntime to populate the grid and lookup array for all experiments: %f",pipeline_totalGPUGridTtotal);	

	printf("\nEND Creating Indexes for each experiment\nMay not use all of them, depending on if neighbortables are reused\n*********************************");


	

	




	double maxSizeNeighborTable=1.5; //in GiB //the maximum size we allow to generate a neighbortable from a previous one (GiB).
									 //if we don't limit this, then the input size may be larger than the GPUs memory
									//and we don't want to batch BOTH the previous neighborTable AND the resultset	
	
	


	double totalGPUTstart=omp_get_wtime();

	unsigned int * totalNeighborsGPU;
	totalNeighborsGPU= new unsigned int[experimentList.size()];
	

	for (int i=0; i<experimentList.size(); i++)
	{
	totalNeighborsGPU[i]=0;		
	}


	//the neighbor tables for each variant
	table * neighborTable[experimentList.size()];
	for (int i=0; i<experimentList.size(); i++)
	{	
	neighborTable[i]=new table[dataPoints.size()];
	}

	//a lookup table that points to the array of neighbors
	//because we cant use vectors on the GPU
	struct gpulookuptable * gpuLookupArray[experimentList.size()];
	for (int i=0; i<experimentList.size(); i++)
	{
	gpuLookupArray[i]= new gpulookuptable[dataPoints.size()];
	}


	

	//first neighbor table:
	//generate the second table from the first table
	//neighbor table:
	// table * neighborTable;
	// neighborTable=new table[dataPoints.size()];
	

	// int * totalNeighborsGPU;
	// totalNeighborsGPU=(int*)malloc(sizeof(int));
	// *totalNeighborsGPU=0;		

	//for nested parallelism for performing dbscan and neighbortable at the same time
	int NUMTHREADS=2;
	omp_set_num_threads(NUMTHREADS);
	int experimentCnt=0;
	std::queue <int> workQueue;
	std::queue <int> freeQueue;
	int experimentID[NUMTHREADS];

	bool workFinished[experimentList.size()];
	for (int i=0; i<experimentList.size(); i++)
	{
		workFinished[i]=false;
	}

	bool usingTable[experimentList.size()];
	for (int i=0; i<experimentList.size(); i++)
	{
		usingTable[i]=true; //assume all experiments can use a previous table
	}


	// bool workReady[experimentList.size()];
	// for (int i=0; i<experimentList.size(); i++)
	// {
	// 	workReady[i]=false;
	// }
		

	double tstartgpudbscan=omp_get_wtime();


	printf("\nGenerating the NeighborTable for the largest epsilon value: %f", experimentList[0].epsilon);	
	double gpuGridTstart=omp_get_wtime();
	makeDistanceTableGPUGridIndexBatches(&dataPoints, &experimentList[0].epsilon, pipeline_index[0], &pipeline_gridMin_x[0], &pipeline_gridMin_y[0], &pipeline_gridNumXCells[0], &pipeline_gridNumYCells[0], pipeline_lookupArr[0], neighborTable[0], &totalNeighborsGPU[0]);
	workQueue.push(0);
	double gpuGridTend=omp_get_wtime();
	printf("\nTime to run the GPU Grid table implementation for the largest epsilon value: %f", gpuGridTend - gpuGridTstart);
	
	printf("\nEnd calling GPU grid kernel");

	



	printf("\n*****************\nGenerating subsequent tables:");
	
	// dbscan(neighborTable[0], dataPoints.size(), experimentList[0].minpts); 
	//printf("\nDBSCAN then Exiting early");
	//return 0;	

	
	printf("\nomp num threads before parallel: %d", omp_get_num_threads());

	#pragma omp parallel num_threads(4)
	{

		int tid=omp_get_thread_num();
		if (tid==0)
		{
			for (int i=1; i<experimentList.size(); i++)
			{
				double sizeOfPreviousTable=(double)sizeof(int)*(totalNeighborsGPU[i-1])/(1024*1024*1024);
				printf("\nSize of previous table: %f", sizeOfPreviousTable);

				if (sizeOfPreviousTable>maxSizeNeighborTable)
				{
					printf("\n\nSize of direct neighbor array: %f (GiB) is too large!\n Going to execute with the index to avoid batching the direct neighbors as well as results\n\n", sizeOfPreviousTable);	
					makeDistanceTableGPUGridIndexBatches(&dataPoints, &experimentList[i].epsilon, pipeline_index[i], &pipeline_gridMin_x[i], &pipeline_gridMin_y[i], &pipeline_gridNumXCells[i], &pipeline_gridNumYCells[i], pipeline_lookupArr[i], neighborTable[i], &totalNeighborsGPU[i]);

					//note that the table from the previous experiment won't be used so it can be freed	
					usingTable[i-1]=false;
				}
				//execute by generating subsequent neighbortable
				else
				{
					printf("\n\n*******************************\n\nGenerating subsequent table: \n");
					

					int * directNeighborArray;
					directNeighborArray=new int[totalNeighborsGPU[i-1]];
					generateNeighborArrayForGPU(dataPoints.size(),neighborTable[i-1], directNeighborArray, gpuLookupArray[i-1]);	
					generateDistanceTableFromPreviousTableBatches(&dataPoints, gpuLookupArray[i-1], directNeighborArray, &totalNeighborsGPU[i-1], &experimentList[i].epsilon, &experimentList[i-1].epsilon, neighborTable[i], &totalNeighborsGPU[i]);
					
					//Note that the table is no longer being used from the previous experiment
					
					usingTable[i-1]=false;	
					
					


				}

				//lets the threads know there's a table to process by DBSCAN
				workQueue.push(i);
			}


		}
		
		//dbscan here (nested parallelism, if/else around the for loop):

		//double tstartGPUdbscanOnly=omp_get_wtime();

		//nested parallelism to take the neighbortable and then dbscan after the table has been created:
		

		else
		{


				printf("\nOuter TID: %d",omp_get_thread_num());
				printf("\nomp num threads: %d", omp_get_num_threads());


				while (experimentCnt<experimentList.size())
				{
					int tid=omp_get_thread_num();
					experimentID[tid]=-1;
					#pragma omp critical
					{
						if(!workQueue.empty())
						{
							experimentID[tid]=workQueue.front();
							workQueue.pop();
							experimentCnt++;
						}
					} //end critical

					//DBSCAN using the table from the experiment 
					if (experimentID[tid]!=-1)
					{
					printf("\nDBSCAN: tid: %d , experiment: %d ",tid,experimentID[tid]);
					
					double dbscanOnlyStart=omp_get_wtime();
					dbscan(neighborTable[experimentID[tid]], dataPoints.size(), experimentList[experimentID[tid]].minpts);
					double dbscanOnlyEnd=omp_get_wtime();	

					
					

					printf("\nTime just for DBSCAN: %f", dbscanOnlyEnd - dbscanOnlyStart);


					//note that dbscan occured and finished
					
					workFinished[experimentID[tid]]=true;
					freeQueue.push(experimentID[tid]);
					
					}


					//while the threads are waiting for work to do.
					//cleaning up neighborTables that can't be used anymore
					
					if (experimentID[tid]==-1)
					{
						#pragma omp critical
						{	
							if (!freeQueue.empty())
							{
								int freeCandidate=freeQueue.front();
								//Check to see if there is a dependancy on the neighborTable from the candidate table to be freed
								//this means that the following experiment has finished DBSCAN, or doesn't need the table because
								//the table was too large, so it needs to produce the table using the index implementation
								//on the GPU
								//if ((usingTable[freeCandidate]==false) && (workFinished[freeCandidate+1]==true) && ((freeCandidate+1) <experimentList.size()))
								if ((usingTable[freeCandidate]==false) && ((freeCandidate+1) <experimentList.size()))
								{
									printf("\nFreeing neighbortable for experiment: %d",freeCandidate);cout.flush();
									delete[] neighborTable[freeCandidate];
									freeQueue.pop();

								}

							} //end if not empty

						} //end critical

					} //end of cleaning up neighbortables
					

				}
		}

	} //end omp parallel section

	//double tendGPUdbscanOnly=omp_get_wtime();

	//printf("\nTime just for DBSCAN: %f", tendGPUdbscanOnly - tstartGPUdbscanOnly);
	

	double tendgpudbscan=omp_get_wtime();

	printf("\nTotal GPU time: %f", tendgpudbscan - tstartgpudbscan);


	/*
	//OLD:
	int * directNeighborArray;
	directNeighborArray=new int[totalNeighborsGPU[0]];

	double tstartneighborarr=omp_get_wtime();
	generateNeighborArrayForGPU(dataPoints.size(),neighborTable[0], directNeighborArray, gpuLookupArray[0]);	
	double tendneighborarr=omp_get_wtime();
	printf("\nTime to create the neighbor array for the GPU: %f", tendneighborarr - tstartneighborarr);


	double tstart2=omp_get_wtime();

	//generateDistanceTableFromPreviousTable(&dataPoints, gpuLookupArray[0], directNeighborArray, &totalNeighborsGPU[0], &experimentList[1].epsilon, neighborTable[1]);
	 generateDistanceTableFromPreviousTableBatches(&dataPoints, gpuLookupArray[0], directNeighborArray, &totalNeighborsGPU[0], &experimentList[1].epsilon, neighborTable[1]);
	double tend2=omp_get_wtime();
	printf("\nTime to create subsequent table for second epsilon value %f: %f",experimentList[0].epsilon,tend2-tstart2);




	double tstartdbscan=omp_get_wtime();
	dbscan(neighborTable[0], dataPoints.size(), experimentList[0].minpts); 
	dbscan(neighborTable[1], dataPoints.size(), experimentList[1].minpts); 
	double tenddbscan=omp_get_wtime();
	printf("\nTime just for DBSCAN: %f", tenddbscan - tstartdbscan);
	
	

	double pipeline_totalGPUTend=omp_get_wtime();

	printf("\nTime excluding the grid construction and lookup array: %f", pipeline_totalGPUTend-pipeline_totalGPUTstart);
	printf("\nGrand total Time gpu dbscan WITH indexing: %f", (pipeline_totalGPUTend - pipeline_totalGPUTstart) + totalGPUGridTtotal);
	*/

	#endif
	

















	///////////////
	//GRID KERNEL GLOBAL MEMORY
	//BATCHING-- DBSCAN WITH A PREVIOUSLY GENERATED NEIGHBOR TABLE
	//ONE TABLE PER DBSCAN VARIANT
	//EITHER GENERATE THE NEIGHBORTABLE USING THE INDEX, OR FROM A PREVIOUS TABLE
	//WITH NEW NEIGHBORTABLES!!!!
	///////////////
	#if SEARCHMODE==7
	printf("\nTHE NEIGHBOR TABLE METHOD WITHOUT VECTORS!!");
	printf("\nCalling GPU grid kernel\n WITH NOSHARED MEMORY KERNEL\n");
	printf("\n***DBSCAN THAT GENERATES A NEIGHBORTABLE FOR EACH VARIANT INSTANCE***\n");

	printf("\n*******\nCreating Indexes for each experiment, although we may not use all of them.\n\n");


	//for nested parallelism for performing dbscan and neighbortable at the same time
	omp_set_num_threads(NTHREADS);


	

	double * pipeline_gridMin_x;
	pipeline_gridMin_x=new double[experimentList.size()];

	double * pipeline_gridMin_y;
	pipeline_gridMin_y=new double[experimentList.size()]; 

	int * pipeline_gridNumXCells;
	pipeline_gridNumXCells= new int [experimentList.size()]; 
	
	int * pipeline_gridNumYCells;
	pipeline_gridNumYCells=new int [experimentList.size()];

	

	//pointers to the grid indexes for each experiment
	//the memory is allocated in the loop below once then umber of cells have been calculated for each experiment
	struct grid ** pipeline_index=new grid*[experimentList.size()];
	
	
	
	//Allocate lookup array
	unsigned int ** pipeline_lookupArr=new unsigned int*[experimentList.size()];
	for (int i=0; i<experimentList.size(); i++)
	{
	pipeline_lookupArr[i]=new unsigned int[dataPoints.size()];
	}
	
	//number of full cells:
	int * pipeline_numFullCells;
	pipeline_numFullCells=new int[experimentList.size()];

	//initialize:
	for (int i=0; i<experimentList.size(); i++)
	{
		pipeline_numFullCells[i]=0;
	}	

	
	

	for (int i=0; i<experimentList.size(); i++)
	{
	printf("\n**************\nCreating index for the experiment epsilon: %f, minpts: %d ",experimentList[i].epsilon,experimentList[i].minpts);	
	//generate grid dimensions:
	generateGridDimensions(&dataPoints, experimentList[i].epsilon, &pipeline_gridMin_x[i], &pipeline_gridMin_y[i], &pipeline_gridNumXCells[i], &pipeline_gridNumYCells[i]);
	
	//allocate memory for the index now that the number of cells have been calculated.
	unsigned int numcells=pipeline_gridNumXCells[i]*pipeline_gridNumYCells[i];
	pipeline_index[i]=new grid[numcells];

	// printf("\n In main: Min x: %f, Min y: %f",gridMin_x,gridMin_y);
	printf("\n Populating indexes for the pipeline: Min x: %f, Min y: %f, Num X cells: %d, Num Y cells: %d",pipeline_gridMin_x[i],pipeline_gridMin_y[i],pipeline_gridNumXCells[i],pipeline_gridNumYCells[i]);
	//populate the index
	populateGridIndexAndLookupArray(&dataPoints, experimentList[i].epsilon, pipeline_index[i], pipeline_lookupArr[i], pipeline_gridMin_x[i], pipeline_gridMin_y[i], pipeline_gridNumXCells[i], pipeline_gridNumYCells[i], &pipeline_numFullCells[i]);
	printf("\nIn main number of full cells: %d", pipeline_numFullCells[i]);cout.flush();
	}

	double pipeline_GPUGridTend=omp_get_wtime();
	


	printf("\nEND Creating Indexes for each experiment\nMay not use all of them, depending on if neighbortables are reused\n*********************************");


	//START TIMER FOR PREAMBLE STUFF (DOESN'T INCLUDE THE INDEX)	
	//MEMORY ALLOCATION FOR ARRAYS ETC. WE ADD THIS TO THE TOTAL TIME AT THE END
	//NEED TO CLEAR THE MEMORY ALLOCATION BETWEEN TRIALS WHICH WE DO NOT INCLUDE


	double preambleTstart=omp_get_wtime();


	




	double maxSizeNeighborTable=1.5; //in GiB //the maximum size we allow to generate a neighbortable from a previous one (GiB).
									 //if we don't limit this, then the input size may be larger than the GPUs memory
									//and we don't want to batch BOTH the previous neighborTable AND the resultset	
	
	

	unsigned int * totalNeighborsGPU;
	totalNeighborsGPU= new unsigned int[experimentList.size()];
	

	for (int i=0; i<experimentList.size(); i++)
	{
	totalNeighborsGPU[i]=0;		
	}


	//the neighbor tables for each variant- each neighbortable points to an array floating somewhere
	//in memory, which are stored in pointersToNeighbors vector (below)
	//the number of arrays for each experiment will be equal to the number of batches to process it.
	neighborTableLookup * neighborTable[experimentList.size()];
	for (int i=0; i<experimentList.size(); i++)
	{	
	neighborTable[i]=new neighborTableLookup[dataPoints.size()];
	}
	//vector of pointers to arrays in memory containing the neighbors within epsilon of each
	//point in the dataset.
	//the struct has the pointer and the size of each array
	std::vector<struct neighborDataPtrs> pointersToNeighbors[experimentList.size()];






	//a lookup table that points to the array of neighbors
	//because we cant use vectors on the GPU
	//only used when constructing a new neighbortable from an old one
	//DEPRICATED: NOW DO IT IN THE MAIN LOOP, SINCE WE DON'T CONSTRUCT THE NEIGHBORTABLE ALL THE TIME
		
	// struct gpulookuptable * gpuLookupArray[experimentList.size()];
	// for (int i=0; i<experimentList.size(); i++)
	// {
	// gpuLookupArray[i]= new gpulookuptable[dataPoints.size()];
	// }


	
	int experimentCnt=0;
	int experimentID[NTHREADS];

	bool workFinished[experimentList.size()];
	for (int i=0; i<experimentList.size(); i++)
	{
		workFinished[i]=false;
	}

	bool usingTable[experimentList.size()];
	for (int i=0; i<experimentList.size(); i++)
	{
		usingTable[i]=true; //assume all experiments can use a previous table
	}


	// bool workReady[experimentList.size()];
	// for (int i=0; i<experimentList.size(); i++)
	// {
	// 	workReady[i]=false;
	// }
		
	double preambleTend=omp_get_wtime();
	printf("\nPREAMBLE time: %f", preambleTend - preambleTstart);


	double trialtimesTotal[NUM_TRIALS]; //The total time
	double trialtimesDBSCAN[NUM_TRIALS]; //Time just to DBSCAN
	
	double trialtimesGPUInitialTable[NUM_TRIALS]; //Time just for GPU Initial Table
	double trialtimesGPUSubsequentTables[NUM_TRIALS]; //Time just for GPU subsequent tables
	double trialtimesGPUTotalTables[NUM_TRIALS]; //Time for initial GPU AND subsequent tables (the addition of the above two)


	for (int x=0; x<NUM_TRIALS; x++)
	{
				printf("\n\n************\nEXECUTING TRIAL NUM: %d\n************\n",x);

				

				////////////////////////////////////////
				//reset variables for next trial:
				////////////////////////////////////////
				std::queue <int> workQueue;
				std::queue <int> freeQueue;	

				for (int j=0; j<experimentList.size(); j++)
				{
					neighborTable[j]=new neighborTableLookup[dataPoints.size()];
					pointersToNeighbors[j].clear();
					totalNeighborsGPU[j]=0;	
					workFinished[j]=false;	
					usingTable[j]=true;
				}
				
				experimentCnt=0;

					

					
				////////////////////////////////////////
				//end reset variables
				////////////////////////////////////////



				//Times not included in the trials: freeing memory for the neighbortables,
				//resetting whether the work has been completed etc.
	

				double tstartTotalTime=omp_get_wtime();


				printf("\nGenerating the NeighborTable for the largest epsilon value: %f", experimentList[0].epsilon);	
				double gpuGridTstart=omp_get_wtime();
				makeDistanceTableGPUGridIndexBatchesAlternateTest(&dataPoints, &experimentList[0].epsilon, pipeline_index[0], &pipeline_gridMin_x[0], &pipeline_gridMin_y[0], &pipeline_gridNumXCells[0], &pipeline_gridNumYCells[0], pipeline_lookupArr[0], neighborTable[0], &pointersToNeighbors[0], &totalNeighborsGPU[0]);
				workQueue.push(0);
				double gpuGridTend=omp_get_wtime();
				printf("\nTime to run the GPU Grid table implementation for the largest epsilon value: %f", gpuGridTend - gpuGridTstart);
				trialtimesGPUInitialTable[x]=gpuGridTend - gpuGridTstart;


				

				
				
				
				
				printf("\n*****************\nStart of generating subsequent tables:");

				
				printf("\nomp num threads before parallel: %d", omp_get_num_threads());

				#pragma omp parallel num_threads(NTHREADS)
				{

					int tid=omp_get_thread_num();
					if (tid==0)
					{

						double subsequentTablesTstart=omp_get_wtime();
						for (int i=1; i<experimentList.size(); i++)
						{
							double sizeOfPreviousTable=(double)sizeof(int)*(totalNeighborsGPU[i-1])/(1024*1024*1024);
							printf("\nSize of previous table: %f", sizeOfPreviousTable);

							if ((sizeOfPreviousTable>maxSizeNeighborTable) || (TABLEFROMPREVIOUS==0))
							{
								printf("\n\nSize of direct neighbor array: %f (GiB) is too large! OR REUSING A TABLE IS DISABLED\n Going to execute with the index to avoid batching the direct neighbors as well as results\n\n", sizeOfPreviousTable);	
								//printf("\nEpsilon of experiment: %f",experimentList[i].epsilon);
								//makeDistanceTableGPUGridIndexBatchesAlternateTest(&dataPoints, &experimentList[i].epsilon, pipeline_index[i], &pipeline_gridMin_x[i], &pipeline_gridMin_y[i], &pipeline_gridNumXCells[i], &pipeline_gridNumYCells[i], pipeline_lookupArr[i], neighborTable[i], &totalNeighborsGPU[i]);
								makeDistanceTableGPUGridIndexBatchesAlternateTest(&dataPoints, &experimentList[i].epsilon, pipeline_index[i], &pipeline_gridMin_x[i], &pipeline_gridMin_y[i], &pipeline_gridNumXCells[i], &pipeline_gridNumYCells[i], pipeline_lookupArr[i], neighborTable[i], &pointersToNeighbors[i], &totalNeighborsGPU[i]);
								//note that the table from the previous experiment won't be used so it can be freed	
								usingTable[i-1]=false;
							}
							//execute by generating subsequent neighbortable from previous one
							#if TABLEFROMPREVIOUS==1
							else
							{
								printf("\n\n*******************************\n\nGenerating subsequent table from previous one: \n");
								
								//The direct neighbors of all of the data points which are pounted to by
								//the struct array below it, gpuLookupArr
								int * directNeighborArray;
								directNeighborArray=new int[totalNeighborsGPU[i-1]];
								//printf("\nAllocating memory for direct neighbor array: %d",totalNeighborsGPU[i-1]);

								struct gpulookuptable * gpuLookupArr;
								gpuLookupArr= new gpulookuptable[dataPoints.size()];
								
								
								generateNeighborArrayForGPUAlternative(dataPoints.size(),neighborTable[i-1], directNeighborArray, gpuLookupArr);	
								generateDistanceTableFromPreviousTableBatchesAlternate(&dataPoints, gpuLookupArr, directNeighborArray, &totalNeighborsGPU[i-1], &experimentList[i].epsilon, &experimentList[i-1].epsilon, neighborTable[i], &pointersToNeighbors[i], &totalNeighborsGPU[i]);
								
								//Note that the table is no longer being used from the previous experiment
								usingTable[i-1]=false;	
								


							}
							//Note that the table is no longer being used from the previous experiment
							usingTable[i-1]=false;
							#endif	
							//lets the threads know there's a table to process by DBSCAN
							workQueue.push(i);
							printf("\nPushing experiment: %d",i);
						}

						double subsequentTablesTend=omp_get_wtime();
						trialtimesGPUSubsequentTables[x]=subsequentTablesTend - subsequentTablesTstart;


					}
					//nested parallelism to take the neighbortable and then dbscan after the table has been created:
					else
					{


							printf("\nOuter TID: %d",omp_get_thread_num());
							printf("\nomp num threads: %d", omp_get_num_threads());

							
							while (experimentCnt<experimentList.size())
							{
								int tid=omp_get_thread_num();
								experimentID[tid]=-1;
								#pragma omp critical
								{
									if(!workQueue.empty())
									{
										experimentID[tid]=workQueue.front();
										workQueue.pop();
										experimentCnt++;
									}
								} //end critical

								//DBSCAN using the table from the experiment 
								if (experimentID[tid]!=-1)
								{
								
								//dbscan(neighborTable[experimentID[tid]], dataPoints.size(), experimentList[experimentID[tid]].minpts);
								double dbscanOnlyStart=omp_get_wtime();
								
								dbscanAlternate(neighborTable[experimentID[tid]], dataPoints.size(), experimentList[experimentID[tid]].minpts);
								
								double dbscanOnlyEnd=omp_get_wtime();	

								printf("\nDBSCAN: tid: %d , experiment: %d, Time just for DBSCAN: %f ",tid,experimentID[tid],dbscanOnlyEnd - dbscanOnlyStart);
								
								//note that dbscan occured and finished
								
								workFinished[experimentID[tid]]=true;
								freeQueue.push(experimentID[tid]);
								
								}


								
								//while the threads are waiting for work to do.
								//cleaning up neighborTables that can't be used anymore
								
								if (experimentID[tid]==-1)
								{
									#pragma omp critical
									{	
										if (!freeQueue.empty())
										{
											int freeCandidate=freeQueue.front();
											//Check to see if there is a dependancy on the neighborTable from the candidate table to be freed
											//this means that the following experiment has finished DBSCAN, or doesn't need the table because
											//the table was too large, so it needs to produce the table using the index implementation
											//on the GPU
											//if ((usingTable[freeCandidate]==false) && (workFinished[freeCandidate+1]==true) && ((freeCandidate+1) <experimentList.size()))
											if ((usingTable[freeCandidate]==false) && ((freeCandidate+1) <experimentList.size()))
											{
												printf("\nFreeing neighbortable for experiment: %d",freeCandidate);cout.flush();
												//delete the struct containing the pointers to the direct neighbors
												delete[] neighborTable[freeCandidate];
												//delete the arrays themselves pointed to by this struct:
												for (int j=0; j<pointersToNeighbors[freeCandidate].size(); j++)
												{
													//free the individual arrays:
													int * ptr= pointersToNeighbors[freeCandidate][j].dataPtr;
													delete[] ptr;
													//delete [] pointersToNeighbors[freeCandidate][j].dataPtr;	
												}

												freeQueue.pop();

											}

										} //end if not empty

									} //end critical

								} //end of cleaning up neighbortables
								

							} //end of while 
							
					
					} //end of else
					

				} //end omp parallel section
				
				double tendTotalTime=omp_get_wtime();

				trialtimesTotal[x]=tendTotalTime - tstartTotalTime;
				
				
				//total time of the GPU:
				trialtimesGPUTotalTables[x]=trialtimesGPUInitialTable[x]+trialtimesGPUSubsequentTables[x];

				//DBSCAN time:
				trialtimesDBSCAN[x]=trialtimesTotal[x]-trialtimesGPUTotalTables[x];

				printf("\nTrial: %d, Total time: %f", x, trialtimesTotal[x]);
				printf("\nTrial: %d, GPU: Initial table: %f",x, trialtimesGPUInitialTable[x]);
				printf("\nTrial: %d, GPU: Subsequent tables:%f",x, trialtimesGPUSubsequentTables[x]);
				printf("\nTrial: %d, GPU: Total tables:%f",x, trialtimesGPUTotalTables[x]);
				printf("\nTrial: %d, CPU: DBSCAN (Total-GPU):%f",x, trialtimesDBSCAN[x]);

				printf("\nOUTPUT STATISTICS ON THE TRIALS, LIKE MIN/MAX FOR SANITY CHECKS");


				//free any remaining neighbortable data that might be around that wasn't freed above
				//so we dont get a memory leak between trials
				
				while(!freeQueue.empty())
				{
					int freeCandidate=freeQueue.front();
					delete[] neighborTable[freeCandidate];

					for (int j=0; j<pointersToNeighbors[freeCandidate].size(); j++)
					{
						//free the individual arrays:
						int * ptr= pointersToNeighbors[freeCandidate][j].dataPtr;
						delete[] ptr;
						//delete [] pointersToNeighbors[freeCandidate][j].dataPtr;	
					}

					freeQueue.pop();
					printf("\nFreed neighbortable for experiment: %d",freeCandidate);

				}	



	} //END TRIALS LOOP	

	
	
	//OUTPUT STATISTICS:
	//SKIP THE FIRST TRIAL THAT WARMS UP THE GPU

	char fname[]="pipeline_stats.txt";
	ofstream pipelineOut;
	pipelineOut.open(fname,ios::app);	


	//averages:
	double trialtimesTotalAvg=0;
	double trialtimesDBSCANAvg=0;

	double trialtimesGPUInitialTableAvg=0;
	double trialtimesGPUSubsequentTablesAvg=0;
	double trialtimesGPUTotalTablesAvg=0;


	//only output if the number of trials is >1 so division by 0 doesn't occur.
	if (NUM_TRIALS>1)
	{
		for (int i=1; i<NUM_TRIALS; i++)
		{
			trialtimesTotalAvg+=trialtimesTotal[i];	
			trialtimesDBSCANAvg+=trialtimesDBSCAN[i];
			trialtimesGPUInitialTableAvg+=trialtimesGPUInitialTable[i];
			trialtimesGPUSubsequentTablesAvg+=trialtimesGPUSubsequentTables[i];
			trialtimesGPUTotalTablesAvg+=trialtimesGPUTotalTables[i];
		}

			trialtimesTotalAvg=trialtimesTotalAvg/((double)NUM_TRIALS-1.0);
			trialtimesDBSCANAvg=trialtimesDBSCANAvg/((double)NUM_TRIALS-1.0);
			trialtimesGPUInitialTableAvg=trialtimesGPUInitialTableAvg/((double)NUM_TRIALS-1.0);
			trialtimesGPUSubsequentTablesAvg=trialtimesGPUSubsequentTablesAvg/((double)NUM_TRIALS-1.0);
			trialtimesGPUTotalTablesAvg=trialtimesGPUTotalTablesAvg/((double)NUM_TRIALS-1.0);

			
			pipelineOut<<"\nTotal time, DBSCAN time (Total-GPU, with multiple experiments DBSCAN overlaps with GPU time. So this isn't quite DBSCAN time.), GPU Tables time, Initial table time, Subsequent table time (if applicable), Allow Table From Previous, NUM THREADS";
			pipelineOut<<endl<<inputFname<<", "<<inputInstanceName;
			pipelineOut<<endl<<std::setprecision(5)<<trialtimesTotalAvg<<", "<<trialtimesDBSCANAvg<<", "<<trialtimesGPUTotalTablesAvg<<", "<<trialtimesGPUInitialTableAvg<<", "<<trialtimesGPUSubsequentTablesAvg<<", "<<TABLEFROMPREVIOUS<<", "<<NTHREADS;

	


	}
	else
	{
		printf("\nStatistics not output, need NUM_TRIALS>1");
	}


	return 0;


	#endif










///////////////
	//GRID KERNEL GLOBAL MEMORY
	//BATCHING-- DBSCAN WITH A PREVIOUSLY GENERATED NEIGHBOR TABLE
	//USES NEW NEIGHBORTABLES
	//SINGLE TABLE WITH ONE EPSILON, MULTIPLE MINPTS IN PARALLEL
	///////////////
	#if SEARCHMODE==8
	printf("\nNEIGHBOR TABLE METHOD WITHOUT VECTORS!!");
	printf("\nCalling GPU grid kernel\n WITH NOSHARED MEMORY KERNEL\n");
	printf("\n***DBSCAN THAT HAS A SINGLE TABLE, FOR ONE EPSILON AND LOTS OF DIFFERENT MINPTS VALUES\n");

	printf("\n*******\nCreating Index\n\n");





	
	
	//OMP
	omp_set_num_threads(NTHREADS);


	

	double * pipeline_gridMin_x;
	pipeline_gridMin_x=new double[1];

	double * pipeline_gridMin_y;
	pipeline_gridMin_y=new double[1]; 

	int * pipeline_gridNumXCells;
	pipeline_gridNumXCells= new int [1]; 
	
	int * pipeline_gridNumYCells;
	pipeline_gridNumYCells=new int [1];

	

	//pointers to the grid indexes for each experiment
	//the memory is allocated in the loop below once then umber of cells have been calculated for each experiment
	struct grid ** pipeline_index=new grid*[1];
	
	//printf("\nExiting early;"); return 0;	cout.flush();
	
	//Allocate lookup array
	// unsigned int ** pipeline_lookupArr=new unsigned int*[1];
	// for (int i=0; i<experimentList.size(); i++)
	// {
	// pipeline_lookupArr[i]=new unsigned int[dataPoints.size()];
	// }

	unsigned int * pipeline_lookupArr=new unsigned int[dataPoints.size()];
	

	//printf("\n**Size of experiment list: %d", (int)experimentList.size());	cout.flush();
	//printf("\nExperiment list epsilon index 0: %f", experimentList[0].epsilon); cout.flush();
	//printf("\nExiting early;"); return 0;	cout.flush();
	
	//number of full cells:
	int * pipeline_numFullCells;
	pipeline_numFullCells=new int[1];

	//initialize:
	for (int i=0; i<1; i++)
	{
		pipeline_numFullCells[i]=0;
	}	



	
	
	
	printf("\n**************\nCreating index for the experiment epsilon: %f",experimentList[0].epsilon);	
	

	//generate grid dimensions:
	generateGridDimensions(&dataPoints, experimentList[0].epsilon, &pipeline_gridMin_x[0], &pipeline_gridMin_y[0], &pipeline_gridNumXCells[0], &pipeline_gridNumYCells[0]);
	
	//allocate memory for the index now that the number of cells have been calculated.
	unsigned int numcells=pipeline_gridNumXCells[0]*pipeline_gridNumYCells[0];
	pipeline_index[0]=new grid[numcells];




	// printf("\n In main: Min x: %f, Min y: %f",gridMin_x,gridMin_y);
	printf("\n Populating indexes for the pipeline: Min x: %f, Min y: %f, Num X cells: %d, Num Y cells: %d",pipeline_gridMin_x[0],pipeline_gridMin_y[0],pipeline_gridNumXCells[0],pipeline_gridNumYCells[0]);
	//populate the index
	populateGridIndexAndLookupArray(&dataPoints, experimentList[0].epsilon, pipeline_index[0], pipeline_lookupArr, pipeline_gridMin_x[0], pipeline_gridMin_y[0], pipeline_gridNumXCells[0], pipeline_gridNumYCells[0], &pipeline_numFullCells[0]);
	printf("\nIn main number of full cells: %d", pipeline_numFullCells[0]);cout.flush();
	

	double pipeline_GPUGridTend=omp_get_wtime();
	


	printf("\nEND Creating Indexes for ONE epsilon value that gets reused for multiple minpts\n*********************************");


	//START TIMER FOR PREAMBLE STUFF (DOESN'T INCLUDE THE INDEX)	
	//MEMORY ALLOCATION FOR ARRAYS ETC. WE ADD THIS TO THE TOTAL TIME AT THE END
	//NEED TO CLEAR THE MEMORY ALLOCATION BETWEEN TRIALS WHICH WE DO NOT INCLUDE


	double preambleTstart=omp_get_wtime();


	





	unsigned int * totalNeighborsGPU;
	totalNeighborsGPU= new unsigned int[1];
	

	for (int i=0; i<1; i++)
	{
	totalNeighborsGPU[i]=0;		
	}


	//the neighbor tables for each variant- each neighbortable points to an array floating somewhere
	//in memory, which are stored in pointersToNeighbors vector (below)
	//the number of arrays for each experiment will be equal to the number of batches to process it.
	neighborTableLookup * neighborTable[1];
	for (int i=0; i<1; i++)
	{	
	neighborTable[i]=new neighborTableLookup[dataPoints.size()];
	}
	//vector of pointers to arrays in memory containing the neighbors within epsilon of each
	//point in the dataset.
	//the struct has the pointer and the size of each array
	std::vector<struct neighborDataPtrs> pointersToNeighbors[1];






	//a lookup table that points to the array of neighbors
	//because we cant use vectors on the GPU
	//only used when constructing a new neighbortable from an old one
	//DEPRICATED: NOW DO IT IN THE MAIN LOOP, SINCE WE DON'T CONSTRUCT THE NEIGHBORTABLE ALL THE TIME
		
	// struct gpulookuptable * gpuLookupArray[experimentList.size()];
	// for (int i=0; i<experimentList.size(); i++)
	// {
	// gpuLookupArray[i]= new gpulookuptable[dataPoints.size()];
	// }


	
	// int experimentCnt=0;
	// int experimentID[NUMTHREADS];

	// bool workFinished[experimentList.size()];
	// for (int i=0; i<experimentList.size(); i++)
	// {
	// 	workFinished[i]=false;
	// }

	// bool usingTable[experimentList.size()];
	// for (int i=0; i<experimentList.size(); i++)
	// {
	// 	usingTable[i]=true; //assume all experiments can use a previous table
	// }


	// bool workReady[experimentList.size()];
	// for (int i=0; i<experimentList.size(); i++)
	// {
	// 	workReady[i]=false;
	// }
		
	double preambleTend=omp_get_wtime();
	printf("\nPREAMBLE time: %f", preambleTend - preambleTstart);


	double trialtimesTotal[NUM_TRIALS]; //The total time
	double trialtimesDBSCAN[NUM_TRIALS]; //Time just to DBSCAN
	double trialtimesGPUInitialTable[NUM_TRIALS]; //Time just for GPU Initial Table

	for (int x=0; x<NUM_TRIALS; x++)
	{
				printf("\n\n************\nEXECUTING TRIAL NUM: %d\n************\n",x);




				//Times not included in the trials: freeing memory for the neighbortables, since
				//this is related to the trials
				neighborTable[0]=new neighborTableLookup[dataPoints.size()];
				pointersToNeighbors[0].clear();
				totalNeighborsGPU[0]=0;	
				

	

				double tstartTotalTime=omp_get_wtime();


				printf("\nGenerating single NeighborTable for the epsilon value: %f", experimentList[0].epsilon);	
				double gpuGridTstart=omp_get_wtime();
				makeDistanceTableGPUGridIndexBatchesAlternateTest(&dataPoints, &experimentList[0].epsilon, pipeline_index[0], &pipeline_gridMin_x[0], &pipeline_gridMin_y[0], &pipeline_gridNumXCells[0], &pipeline_gridNumYCells[0], pipeline_lookupArr, neighborTable[0], &pointersToNeighbors[0], &totalNeighborsGPU[0]);
				//workQueue.push(0);
				double gpuGridTend=omp_get_wtime();
				printf("\nTime to run the GPU Grid table implementation for the epsilon value common to all DBSCAN instances: %f", gpuGridTend - gpuGridTstart);
				trialtimesGPUInitialTable[x]=gpuGridTend - gpuGridTstart;

				//DBSCAN ALL VARIANTS:
				#pragma omp parallel for num_threads(NTHREADS) schedule(static,1)
				for (int i=0; i<experimentList.size(); i++)
				{
				int tid=omp_get_thread_num();	
				printf("\nDBSCAN experiment: %d, tid: %d",i,tid);	
				dbscanAlternate(neighborTable[0], dataPoints.size(), experimentList[i].minpts);
				}
				
				double tendTotalTime=omp_get_wtime();

				//end total time
				trialtimesTotal[x]=tendTotalTime - tstartTotalTime;

				//DBSCAN time:
				trialtimesDBSCAN[x]=trialtimesTotal[x]-trialtimesGPUInitialTable[x];

				printf("\nTrial: %d, Total time: %f", x, trialtimesTotal[x]);
				printf("\nTrial: %d, GPU: Initial table: %f",x, trialtimesGPUInitialTable[x]);
				printf("\nTrial: %d, CPU: DBSCAN (Total-GPU):%f",x, trialtimesDBSCAN[x]);

				//free memory between trials so we don't get a memory leak:
				//But don't time this part since its related to the trials
				for (int j=0; j<pointersToNeighbors[0].size(); j++)
				{
					//free individual arrays
					int * ptr=pointersToNeighbors[0][j].dataPtr;
					delete[] ptr;
					

				}
				pointersToNeighbors[0].clear();


				// for (int j=0; j<pointersToNeighbors[freeCandidate].size(); j++)
				// 	{
				// 		//free the individual arrays:
				// 		int * ptr= pointersToNeighbors[freeCandidate][j].dataPtr;
				// 		delete[] ptr;
				// 		//delete [] pointersToNeighbors[freeCandidate][j].dataPtr;	
				// 	}
				
				

	} //END TRIALS LOOP	




	//OUTPUT STATISTICS:
	//SKIP THE FIRST TRIAL THAT WARMS UP THE GPU

	char fname[]="reuse_single_table_stats.txt";
	ofstream reuseOut;
	reuseOut.open(fname,ios::app);	


	//averages:
	double trialtimesTotalAvg=0;
	double trialtimesDBSCANAvg=0;

	double trialtimesGPUInitialTableAvg=0;
	
	


	//only output if the number of trials is >1 so division by 0 doesn't occur.
	if (NUM_TRIALS>1)
	{
		for (int i=1; i<NUM_TRIALS; i++)
		{
			trialtimesTotalAvg+=trialtimesTotal[i];	
			trialtimesDBSCANAvg+=trialtimesDBSCAN[i];
			trialtimesGPUInitialTableAvg+=trialtimesGPUInitialTable[i];
			
		}

			trialtimesTotalAvg=trialtimesTotalAvg/((double)NUM_TRIALS-1.0);
			trialtimesDBSCANAvg=trialtimesDBSCANAvg/((double)NUM_TRIALS-1.0);
			trialtimesGPUInitialTableAvg=trialtimesGPUInitialTableAvg/((double)NUM_TRIALS-1.0);
			

			
			reuseOut<<"\nTotal time, DBSCAN time, Initial table time, NUM THREADS";
			reuseOut<<endl<<inputFname<<", "<<inputInstanceName;
			reuseOut<<endl<<std::setprecision(5)<<trialtimesTotalAvg<<", "<<trialtimesDBSCANAvg<<", "<<trialtimesGPUInitialTableAvg<<", "<<NTHREADS;

	

	}
	else
	{
		printf("\nStatistics not output, need NUM_TRIALS>1");
	}



	
	

	return 0;


	#endif










	/*


	///////////////
	//GPU implementation
	///////////////

	#if SEARCHMODE==5
	///////////////////////////////////
	//GPU kernel variables	
	///////////////////////////////////

	


	
	

	
	


	//to create a neighbor table on the GPU using a higher epsilon for a lower epsilon:
	
	//an array that contains all of the neighbors for each point.
	//There is an array of structures that point to this array that denote the indicies in the array
	//that contains the neighbors of each point  
	int * directNeighborArray;
	directNeighborArray=new int[*totalNeighbors];
	
	//a lookup table that points to the array of neighbors above
	//because we cant use vectors on the GPU
	struct gpulookuptable * gpuLookupArray;
	gpuLookupArray= new gpulookuptable[dataPoints.size()];
	generateNeighborArrayForGPU(dataPoints.size(),neighborTable, directNeighborArray, gpuLookupArray);
	
	//makeDistanceTableFromPreviousTable();
	

	//print GPU lookup[ array]
	// printf("\nGPU lookup array *******");
	// for (int i=0; i<dataPoints.size(); i++)
	// {
	// 	printf("\nPoint id: %d In distance: ",i);
	// 	for (int j=gpuLookupArray[i].indexmin; j<=gpuLookupArray[i].indexmax; j++)
	// 	{
	// 		printf("%d, ", directNeighborArray[j]);
	// 	}
	// }

	///////////////////////////////////////////
	//GENERATING ONE TABLE FROM ANOTHER ON THE GPU
	///////////////////////////////////////////
	//second neighbor table:
	//generate the second table from the first table
	//neighbor table:
	table * neighborTable2;
	neighborTable2=new table[dataPoints.size()];


	double * epsilon2;
	epsilon2=(double*)malloc(sizeof(double));
	*epsilon2=*epsilon/2.0;

	double tstart2=omp_get_wtime();
	generateDistanceTableFromPreviousTable(&dataPoints, gpuLookupArray, directNeighborArray, totalNeighbors, epsilon2, neighborTable2);
	double tend2=omp_get_wtime();
	printf("\nTime to create subsequent table for epsilon value %f: %f",*epsilon2,tend2-tstart2);

	//////////////////////////////////



	//call DBSCAN (from first table- neighborTable):
	double dbscantstart=omp_get_wtime();
	dbscan(neighborTable, dataPoints.size(), 5); //5 is minpts
	double dbscantend=omp_get_wtime();
	printf("\ntime to dbscan: %f",dbscantend-dbscantstart);

	printf("\ntotal time for GPU implementation: %f",(tend-tstart)+(dbscantend-dbscantstart));
	



	//***************************
	//Grid implementation
	
	//void generateGrid(std::vector<struct dataElem>* dataPoints, double epsilon, struct grid * index, int * dataLookup, double * gridMin_x, double * gridMin_y, int * gridNumXCells, int * gridNumYCells)
	//struct grid * index; //allocate the grid cells and index min and max in generate grid
	//int * dataLookupArray; //allocate lookup array in generate grid
	
	

	


	// //given a point, get the ids of the adjacent grid cells that contain points
	// int * gridIDs; //these grid ids contain points
	// gridIDs=new int[9];
	// int numAdjCells=0; //the number of adjacent cells from a grid cell that contain points


	// //calcLinearID(&dataPoints[0], index, *epsilon, gridMin_x, gridMin_y, gridNumXCells, gridNumYCells, gridIDs, &numAdjCells);
	// //printf("\nlinear id: %d",id);

	// //test looking up all of the adjacent cell IDs for each point on the CPU:

	// for (int i=0; i<dataPoints.size(); i++)
	// {
	// 	calcLinearID(&dataPoints[i], index, *epsilon, gridMin_x, gridMin_y, gridNumXCells, gridNumYCells, gridIDs, &numAdjCells);
	// 	printf("\ndata point %d, num grid ids: %d, adj cells: ",i, numAdjCells);
	// 	for (int j=0; j<numAdjCells; j++)
	// 	{
	// 		printf("%d, ", gridIDs[j]);
	// 	}
	// }	

	

	
	printf("\n***************************************");

	////////////////////////////////////////////////
	//testing using key value pairs for the results instead of a struct
	//temp limit scope
	{

	int * totalNeighborsGPU;
	totalNeighborsGPU=(int*)malloc(sizeof(int));
	*totalNeighborsGPU=0;		

	table * neighborTable3;
	neighborTable3=new table[dataPoints.size()];



	


	printf("\nCalling GPU grid kernel\n WITH NOSHARED MEMORY KERNEL \nTESTING KEY VALUE\n");
	printf("\nKEY VALUE IMPLEMENTED! NOW TESTING BATCHED EXECUTION\n\n");
	double totalGPUTstart=omp_get_wtime();
	
	
	double gpuGridTstart=omp_get_wtime();
	//original:
	// makeDistanceTableGPUGridIndex(&dataPoints,epsilon, index, &gridMin_x, &gridMin_y, &gridNumXCells, &gridNumYCells, lookupArr, neighborTable3, totalNeighborsGPU);
	
	//with batches:
	makeDistanceTableGPUGridIndexTestBatches(&dataPoints,epsilon, index, &gridMin_x, &gridMin_y, &gridNumXCells, &gridNumYCells, lookupArr, neighborTable3, totalNeighborsGPU);
	
	double gpuGridTend=omp_get_wtime();
	
	printf("\ntime to run the GPU Grid table implementation: %f", gpuGridTend - gpuGridTstart);
	
	printf("\nEnd calling GPU grid kernel");

	// printf("\nEXITING early...");
	// return 0;

	double dbscantstartgpuindex=omp_get_wtime();
	dbscan(neighborTable3, dataPoints.size(), *minpts); //5 is minpts
	double dbscantendgpuindex=omp_get_wtime();
	printf("\ntime to dbscan: %f",dbscantendgpuindex-dbscantstartgpuindex);

	double totalGPUTend=omp_get_wtime();

	printf("\ntime excluding the grid construction and lookup array: %f", totalGPUTend-totalGPUTstart);
	printf("\ngrand total time gpu dbscan with indexing: %f", (totalGPUTend - totalGPUTstart) + totalGPUGridTtotal);


	// printf("\nstarting sleep");
	// #pragma omp barrier

	// system("sleep 3");
	// printf("\nending sleep");

	}

	//////////////////////////////////







	


	#endif
	*/


	#if SEARCHMODE==9


	//GPU test using the grid index to generate the table
	//each grid cell is processed by a block
	//this one is DATA AWARE: we pass in the amount of shared memory required
	
	
		
	printf("\nCalling GPU grid kernel With Shared Memory\n DATA AWARE\nEach non-empty grid cell processed by a block!\n");
	printf("\nNOT VALIDATED, UNFINISHED DUE TO NEEDING TO ACCOMODATE CELLS WITH MORE SHARED MEMORY THAN POSSIBLE TO ALLOCATE!\n");



	double totalGPUTstart=omp_get_wtime();
	
	int * totalNeighborsGPU;
	totalNeighborsGPU=(int*)malloc(sizeof(int));
	*totalNeighborsGPU=0;

	table * neighborTable;
	neighborTable=new table[dataPoints.size()];


	unsigned int maxDataPoints=calcMaxSharedMemDataAware(index, gridNumXCells, gridNumYCells);
	printf("\nThe maximum number of data points overlapping adjacent grid cells is: %d", maxDataPoints);


	
	double gpuGridTstart=omp_get_wtime();

	makeDistanceTableGPUGridIndexWithSMBlockDataAware(&dataPoints,epsilon, index, &gridMin_x, &gridMin_y, &gridNumXCells, &gridNumYCells, lookupArr, neighborTable, totalNeighborsGPU, maxDataPoints);
	double gpuGridTend=omp_get_wtime();
	
	printf("\nTime to run the GPU Grid table implementation: %f", gpuGridTend - gpuGridTstart);
	
	printf("\nEnd calling GPU grid kernel");

	//printf("\nRETURNING EARLY IN MAIN!!!!");
	//return 0;


	double dbscantstartgpuindex=omp_get_wtime();
	dbscan(neighborTable, dataPoints.size(), *minpts); //5 is minpts
	double dbscantendgpuindex=omp_get_wtime();
	printf("\nTime to dbscan: %f",dbscantendgpuindex-dbscantstartgpuindex);

	double totalGPUTend=omp_get_wtime();

	printf("\nTime excluding the grid construction and lookup array: %f", totalGPUTend-totalGPUTstart);
	printf("\nGrand total Time gpu dbscan with indexing: %f", (totalGPUTend - totalGPUTstart) + totalGPUGridTtotal);
	
	
	
	



	#endif



	//TEST THE TOTAL COUNTS OF NEIGHBORS IN CPU PROTOTYPE IMPLEMENTATIONS OF
	//GRIDS VS. BRUTE FORCE

	#if SEARCHMODE==10

	//TESTING CPU VERSION OF THE GRID VS CPU BRUTE FORCE:
	printf("\nPROTOTYPE CPU GRID VS CPU BRUTE FORCE TOTAL NEIGHBORS -- MAY NOT WORK");
	printf("\n*******GRID CPU");

	int * gridIDs; //these grid ids contain points
	gridIDs=new int[9];
	int numAdjCells=0; //the number of adjacent cells from a grid cell that contain points

	unsigned int cpucnt=0;

	std::vector<int> tmpInDist;

	double tstartcpu=omp_get_wtime();	
	for (int i=0; i<dataPoints.size(); i++)
	{
		double x1=dataPoints[i].x;
		double y1=dataPoints[i].y;

		printf("\nPoint id: %d In distance: ",i);
		
		calcLinearID(&dataPoints[i], index, *epsilon, gridMin_x, gridMin_y, gridNumXCells, gridNumYCells, gridIDs, &numAdjCells);

		tmpInDist.clear();

		for (int j=0; j<numAdjCells; j++)
		{
			int gridID=gridIDs[j];

			
			for (int k=index[gridID].indexmin; k<=index[gridID].indexmax; k++)
			{
				int elemid=lookupArr[k];
				double x2=dataPoints[elemid].x;
				double y2=dataPoints[elemid].y;	

				//XXXXXXXXXXXXXX
				//STORE AND SORT THE ONES IN THE DISTANCE
				//XXXXXXXXXXXXXX

					if (sqrt(((x1-x2)*(x1-x2))+((y1-y2)*(y1-y2)))<=(*epsilon))
					{
						//printf("%d, ",elemid);
						tmpInDist.push_back(elemid);
						cpucnt++;
					}
			}

		}

		//print the ones in distance after sorted
		std::sort(tmpInDist.begin(),tmpInDist.end());
		for (int l=0; l<tmpInDist.size(); l++)
		{
			printf("%d, ", tmpInDist[l]);
		}

	}
	double tendcpu=omp_get_wtime();

	printf("\nGrid CPU: count within epsilon: %d", cpucnt);
	printf("\nGrid CPU time: %f",tendcpu-tstartcpu);
	



	//test comparison vs CPU brute force:
	


	
	printf("\n*******CPU BRUTE FORCE");

	unsigned int cpucnt2=0;

	double tstartcpu2=omp_get_wtime();	
	for (int i=0; i<dataPoints.size(); i++)
	{
		double x1=dataPoints[i].x;
		double y1=dataPoints[i].y;

		printf("\nPoint id: %d In distance: ",i);
		
		for (int j=0; j<dataPoints.size(); j++)
		{
			
			double x2=dataPoints[j].x;
			double y2=dataPoints[j].y;
			if (sqrt(((x1-x2)*(x1-x2))+((y1-y2)*(y1-y2)))<=(*epsilon))
			{
				printf("%d, ",j);
				cpucnt2++;
			}
		}
	}
	double tendcpu2=omp_get_wtime();

	printf("\nCPU: count within epsilon: %d", cpucnt2);
	printf("\nCPU time: %f",tendcpu2-tstartcpu2);
	
		
	

	////////////////////////////



	#endif
	



	//////////////////////////
	//END GPU IMPLEMENTATION
	/////////////////////////



		



	printf("\n\n\n");
	return 0;
}





//prototype CPU implementation.

//some of it to be ported to GPU:

//calculates adjacent cell ids from a point
void calcLinearID(struct dataElem * point, struct grid * index, double epsilon, double gridMin_x, double gridMin_y, int gridNumXCells, int gridNumYCells, int * outGridCellIDs, int * outNumGridCells)
{
	int xCellID=(point->x-gridMin_x)/epsilon;
	int yCellID=(point->y-gridMin_y)/epsilon;

	
	int minXCellID=0;
	int maxXCellID=0;
	int minYCellID=0;
	int maxYCellID=0;
	

	//calculate the min and max x and y cell ids by adding and subtracting one from each value
	//deal with exception cases below.

	minXCellID=max(0,xCellID-1);
	maxXCellID=min(xCellID+1,gridNumXCells-1);
	minYCellID=max(0,yCellID-1);
	maxYCellID=min(yCellID+1,gridNumYCells-1);


	//enumerate the cells in 2D, then convert into 1D
	//only store the cells that have data in them

	//IN CUDA DO #pragma unroll on both loops
	int cnt=0;
	for (int i=minYCellID; i<=maxYCellID; i++)
		{
		for (int j=minXCellID; j<=maxXCellID; j++){
			int linearID=(i*gridNumXCells)+j;			
			
			if(index[linearID].indexmin!=-1) 
			{
				outGridCellIDs[cnt]=linearID;
				cnt++;
			} 
		}
	}

	*outNumGridCells=cnt;
}


void populateGridIndexAndLookupArray(std::vector<struct dataElem>* dataPoints, double epsilon, struct grid * index, unsigned int * lookupArr, double gridMin_x, double gridMin_y, int gridNumXCells, int gridNumYCells, int * numFullCells)
{

	/////////////////////////////////
	//Populate grid lookup array
	//and corresponding indicies in the lookup array
	/////////////////////////////////

	printf("\nSize of dataset: %lu", dataPoints->size());

	//contains the indices of the data points in the database that are in each grid cell.
	//int * cellMemberArray=new int[dataPoints->size()];



	//Temp vector that stores the data points inside each cell that are based on the linearized cell IDs (1 dimensional)
	//std::vector<int> gridElemIDs[gridNumXCells*gridNumYCells];

	unsigned int totalCells=(gridNumXCells*gridNumYCells);

	std::vector<unsigned int> * gridElemIDs;
	gridElemIDs = new std::vector<unsigned int>[totalCells];

	

	for (int i=0; i<dataPoints->size(); i++)
	{
		//calculate the linearized cell id in row-major order from the x and y values of the point
		int xCellID=((*dataPoints)[i].x-gridMin_x)/epsilon;
		int yCellID=((*dataPoints)[i].y-gridMin_y)/epsilon;

		unsigned int linearID=(yCellID*gridNumXCells)+xCellID;

		if (linearID > totalCells)
		{
			printf("\n\nERROR Linear ID is: %d\n\n", linearID);
		}

		//printf("\nX cell id: %d, Y cell id: %d, linear ID: %d",xCellID,yCellID,linearID);
		gridElemIDs[linearID].push_back(i);
	}




	

	int cnt=0;
	int cntEmptyCells=0;
	int cntFullCells=0;

	//populate index and lookup array
	for (int i=0; i<totalCells; i++)
	{

		if(gridElemIDs[i].size()!=0)
		{	
			index[i].indexmin=cnt;
			cntFullCells++;
			for (int j=0; j<gridElemIDs[i].size(); j++)
			{
				if (j>((dataPoints->size()-1)))
				{
					printf("\n\n***ERROR Value of a data point is larger than the dataset! %d\n\n", j);
					return;
				}
				lookupArr[cnt]=gridElemIDs[i][j]; //problem is with this line, is it i or j?
				cnt++;
			}
			index[i].indexmax=cnt-1;
		}

		else
		{
			index[i].indexmin=-1;
			index[i].indexmax=-1;			
			cntEmptyCells++;
		}
	}

	// printf("\nExiting grid populate method early!");
	// return;

	printf("\nFull cells: %d (%f, fraction full)",cntFullCells, cntFullCells/double(totalCells));
	printf("\nEmpty cells: %d (%f, fraction empty)",cntEmptyCells, cntEmptyCells/double(totalCells));

	*numFullCells=cntFullCells;


	printf("\nSize of index to be sent to GPU (GiB): %f", (double)sizeof(struct grid)*(totalCells)/(1024.0*1024.0*1024.0));


	//print for testing
	// int count=0;
	// for (int i=0; i<gridNumXCells*gridNumYCells; i++)
	// {
		
	// 	printf("\nLinear id: %d, index min: %d, index max: %d \nids: ", i, index[i].indexmin, index[i].indexmax);
	// 	if (index[i].indexmin!=-1 && index[i].indexmax!=-1)
	// 	{
	// 		for (int j=index[i].indexmin; j<=index[i].indexmax; j++)
	// 		{
	// 			count++;
	// 			printf("%d, ",lookupArr[j]);
	// 		}
	// 	}

	// }

	// printf("\ntest number of data elems: %d", count);





	/////////////////////////////////
	//END Populate grid lookup array
	//and corresponding indicies in the lookup array
	/////////////////////////////////


}


void generateGridDimensions(std::vector<struct dataElem>* dataPoints, double epsilon, double * gridMin_x, double * gridMin_y, int * gridNumXCells, int * gridNumYCells)
{

	printf("\nGenerating grid dimensions.");
	
		

	/////////////////////////////////
	//calculate the min and max points in the x and y dimension
	/////////////////////////////////
	double minPnt_x=(*dataPoints)[0].x;
	double maxPnt_x=(*dataPoints)[0].x;
	double minPnt_y=(*dataPoints)[0].y;
	double maxPnt_y=(*dataPoints)[0].y;





	for (int i=1; i<dataPoints->size(); i++)
	{
		if ((*dataPoints)[i].x<minPnt_x)
		{
			minPnt_x=(*dataPoints)[i].x;
		}

		if ((*dataPoints)[i].x>maxPnt_x)
		{
			maxPnt_x=(*dataPoints)[i].x;
		}

		if ((*dataPoints)[i].y<minPnt_y)
		{
			minPnt_y=(*dataPoints)[i].y;
		}

		if ((*dataPoints)[i].y>maxPnt_y)
		{
			maxPnt_y=(*dataPoints)[i].y;
		}

	}

	printf("\nGrid: Points in dataset: \nMin x,y: %f, %f", minPnt_x, minPnt_y);
	printf("\nGrid: Points in dataset: \nMax x,y: %f, %f", maxPnt_x, maxPnt_y);


	/////////////////////////////////
	//end calculate the min and max points in the x and y dimension
	/////////////////////////////////


	/////////////////////////////////
	//Calculate the start of the grid along the x and y dimensions
	/////////////////////////////////

	//The cell width is epsilon, such that we only check the neighbors
	//The starting point of the cell in the grid is going to be the min points in the
	//x and y dimensions subtract 1.1 epsilon to avoid possible boundary conditions

	
	minPnt_x=minPnt_x-(0.5*epsilon);
	minPnt_y=minPnt_y-(0.5*epsilon);



	//pass out of the function
	*gridMin_x=minPnt_x;
	*gridMin_y=minPnt_y;

	

	//total spatial extent in the x and y dimensions
	double xWidth=maxPnt_x-minPnt_x;
	double yWidth=maxPnt_y-minPnt_y;

	//total cells in x and y dimensions:
	int xCells=ceil(xWidth/epsilon);
	int yCells=ceil(yWidth/epsilon);

	//pass out of the function
	*gridNumXCells=xCells;
	*gridNumYCells=yCells;

	printf("\nGrid: Total x cells, y cells: %d, %d",xCells,yCells);




	/////////////////////////////////
	//End calculate start of the grid along x and y dimensions
	/////////////////////////////////




}


//FOR THE DATA-AWARE IMPLEMENTATION
//CALCULATE THE MAXIMUM AMOUNT OF SHARED MEMORY REQUIRED FOR THE OVERLAPPING
//DATA ELEMENTS IN THE CELLS
unsigned int calcMaxSharedMemDataAware(struct grid * index, int gridNumXCells, int gridNumYCells)
{

	int totalGridCells=gridNumXCells*gridNumYCells;
	

	unsigned int maxOverlappingPoints=0;


	for (int i=0; i<totalGridCells; i++)
	{

	int xCellID=i%gridNumXCells;
	int yCellID=i/gridNumXCells;

	

	int minXCellID=0;
	int maxXCellID=0;
	int minYCellID=0;
	int maxYCellID=0;
	

	//calculate the min and max x and y cell ids by adding and subtracting one from each value
	//deal with exception cases below.

	minXCellID=max(0,xCellID-1);
	maxXCellID=min(xCellID+1,gridNumXCells-1);
	minYCellID=max(0,yCellID-1);
	maxYCellID=min(yCellID+1,gridNumYCells-1);

	
	//enumerate the cells in 2D, then convert into 1D
	//only store the cells that have data in them

	
	int tmpCntPoints=0;
	
	for (int i=minYCellID; i<=maxYCellID; i++){
		for (int j=minXCellID; j<=maxXCellID; j++){
			int linearID=(i*gridNumXCells)+j;			
			
			if(index[linearID].indexmin!=-1) 
			{
				tmpCntPoints+=index[linearID].indexmax-index[linearID].indexmin+1;
			} 
		}
	}
	
	if (tmpCntPoints>maxOverlappingPoints)
	{
		maxOverlappingPoints=tmpCntPoints;
	}


	} //end outer for loop


	return maxOverlappingPoints;

} //end of function









/*
//multiple points per MBB:
//called this a multiple point box (MPB)
void createEntryMBBMultiplePoints(std::vector<dataElem> *dataPoints, std::vector<std::vector<int> > *MPB_ids, MPBRect * dataRectsMPB)
{
	int MPB_cnt=0;
	int dataPointCnt=0;
	for (int i=0; i<(*dataPoints).size(); i+=MBBSIZE){
		
		//create new space in the MSB vector	
		(*MPB_ids).push_back(vector<int>());
		for (int j=0; j<MBBSIZE; j++)
		{



			//don't want to go over the size of the number of datapoints.	
			//the last MBB might have less than MBBSIZE number of points in it	
			//insert the dataElem ID into the vector of vectors
			if (((MPB_cnt*MBBSIZE)+j)<(*dataPoints).size()){	
			(*MPB_ids)[MPB_cnt].push_back((MPB_cnt*MBBSIZE)+j);
			
			//printf("\nMBB count: %d, data point cnt: %d",MPB_cnt, dataPointCnt);
			dataPointCnt++;
			}
		
		}
		
		
		//create a new MBB for the point(s)
		//&(*MPB_ids)[MPB_cnt] is confusing, but it's a pointer to a vector inside the vector that stores the IDs
		//of the data points that are within each MPB
		dataRectsMPB[MPB_cnt].CreateMBB(dataPoints, &(*MPB_ids)[MPB_cnt]);
		

		//printf("\nMBB min: %f,%f,%f,%f, MBB max: %f,%f,%f,%f",dataRectsMPB[MPB_cnt].MBB_min[0],dataRectsMPB[MPB_cnt].MBB_min[1],dataRectsMPB[MPB_cnt].MBB_min[2],dataRectsMPB[MPB_cnt].MBB_min[3],dataRectsMPB[MPB_cnt].MBB_max[0],dataRectsMPB[MPB_cnt].MBB_max[1],dataRectsMPB[MPB_cnt].MBB_max[2],dataRectsMPB[MPB_cnt].MBB_max[3]);
		MPB_cnt++;
	} //end main for loop


	//testing MSBs:
	// for (int i=0; i<(*MSB_ids).size();i++)
	// {
	// 	printf("\nMBB num: %d", i);
	// 	printf("\nMBB dims: min: %f,%f,%f,%f max: %f,%f,%f,%f", dataRectsMSB[i].MBB_min[0],dataRectsMSB[i].MBB_min[1],dataRectsMSB[i].MBB_min[2],dataRectsMSB[i].MBB_min[3],dataRectsMSB[i].MBB_max[0],dataRectsMSB[i].MBB_max[1],dataRectsMSB[i].MBB_max[2],dataRectsMSB[i].MBB_max[3]);
	// 	for (int j=0; j<(*MSB_ids)[i].size();j++)
	// 	{
	// 		int pntid=(*MSB_ids)[i][j];
	// 		printf("\npoint id: %d, point: %f, %f, %f, %f", pntid, (*dataPoints)[pntid].x,(*dataPoints)[pntid].y,(*dataPoints)[pntid].val,(*dataPoints)[pntid].time);
	// 	}
	// }

	printf("\ninserted this many points into the MPBs: %d",dataPointCnt);
	printf("\n Total MPBs created: %d", MPB_cnt);

}	





//create MBBs for R-tree
void createEntryMBBs(std::vector<dataElem> *dataPoints, Rect * dataRects)
{
	for (int i=0; i<(*dataPoints).size(); i++){
		dataRects[i].P1[0]=(*dataPoints)[i].x;
		dataRects[i].P1[1]=(*dataPoints)[i].y;
		// dataRects[i].P1[2]=(*dataPoints)[i].val;
		// dataRects[i].P1[3]=(*dataPoints)[i].time;
		dataRects[i].pid=i;
		dataRects[i].CreateMBB();
	}

}	

void BruteForceFindPoints(std::vector<dataElem> *dataPoints, std::vector<int> *candidateSet)
{
	for (int i=0; i<(*dataPoints).size();i++){

		if((*dataPoints)[i].val>80 && (*dataPoints)[i].x>160 && (*dataPoints)[i].x<180 && (*dataPoints)[i].y>10 && (*dataPoints)[i].y<20)
		{
			(*candidateSet).push_back(i);
		}

	}
}



//create Query MBBs for R-tree
*/



